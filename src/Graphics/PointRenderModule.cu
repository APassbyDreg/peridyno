#include "hip/hip_runtime.h"
#pragma once

#include <GL/glew.h>
#include "PointRenderModule.h"
#include "Topology/PointSet.h"
#include "Vector.h"
#include "Framework/Node.h"
#include "OpenGLContext.h"
#include "Color.h"


namespace dyno
{
	IMPLEMENT_CLASS(PointRenderModule)

	PointRenderModule::PointRenderModule()
		: VisualModule()
		, m_color(Vector3f(0.8, 0.8, 0.8))
	{
		m_minIndex.setValue(0);
		m_maxIndex.setValue(1);

		m_refV = m_minIndex.getValue();

		this->attachField(&m_minIndex, "minIndex", "minIndex", false);
		this->attachField(&m_maxIndex, "maxIndex", "maxIndex", false);

		this->attachField(&m_vecIndex, "vectorIndex", "vectorIndex", false);
		this->attachField(&m_scalarIndex, "scalarIndex", "scalarIndex", false);
	}

	PointRenderModule::~PointRenderModule()
	{
	}

	bool PointRenderModule::initializeImpl()
	{
		m_pointRender = std::make_shared<PointRender>();

		Log::sendMessage(Log::Info, "PointRenderModule successfully initialized!");

		return true;
	}

	__global__ void PRM_MappingColor(
		DArray<glm::vec3> color,
		DArray<Vector3f> index,
		float minIndex,
		float maxIndex)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= color.size()) return;

		float index_i = index[tId].norm();

		index_i = index_i > maxIndex ? maxIndex : index_i;
		index_i = index_i < minIndex ? minIndex : index_i;

		float a = (index_i - minIndex) / (maxIndex - minIndex);

		Color hsv;
		hsv.HSVtoRGB(240, 1-a, 1);

		color[tId] = glm::vec3(hsv.r, hsv.g, hsv.b);
	}

	__global__ void PRM_MappingColor(
		DArray<glm::vec3> color,
		DArray<float> index,
		float refV,
		float minIndex,
		float maxIndex)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= color.size()) return;

		float index_i = index[tId];

		index_i = index_i > maxIndex ? maxIndex : index_i;
		index_i = index_i < minIndex ? minIndex : index_i;

		float a = (index_i - refV) / (maxIndex - minIndex);

		Color hsv;
		hsv.HSVtoRGB(a * 120 + 120, 1, 1);

		color[tId] = glm::vec3(hsv.r, hsv.g, hsv.b);
	}

	void PointRenderModule::updateRenderingContext()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return;
		}

		auto pSet = TypeInfo::cast<PointSet<DataType3f>>(parent->getTopologyModule());
		if (pSet == nullptr)
		{
			Log::sendMessage(Log::Error, "PointRenderModule: The topology module is not supported!");
			return;
		}

		DArray<float3>* xyz = (DArray<float3>*)&(pSet->getPoints());
		
		if (xyz->size() != m_pointRender->numOfPoints())
		{
			m_pointRender->resize(xyz->size());
			m_colorArray.resize(xyz->size());
		}
		

		if (!m_vecIndex.isEmpty())
		{
			uint pDims = cudaGridSize(xyz->size(), BLOCK_SIZE);
			PRM_MappingColor << <pDims, BLOCK_SIZE >> > (
				m_colorArray,
				m_vecIndex.getValue(),
				m_minIndex.getValue(),
				m_maxIndex.getValue());
			cuSynchronize();

			m_pointRender->setColor(m_colorArray);
		}
		else if (!m_scalarIndex.isEmpty())
		{
			uint pDims = cudaGridSize(xyz->size(), BLOCK_SIZE);
			PRM_MappingColor << <pDims, BLOCK_SIZE >> > (
				m_colorArray,
				m_scalarIndex.getValue(),
				m_refV,
				m_minIndex.getValue(),
				m_maxIndex.getValue());
			cuSynchronize();

			m_pointRender->setColor(m_colorArray);
		}
		else
		{
			m_pointRender->setColor(glm::vec3(m_color[0], m_color[1], m_color[2]));
		}

		
		//if (m_colorArray)
		//	m_pointRender->setColorArray(*(DArray<float3>*)m_colorArray.get());
		
		m_pointRender->setVertexArray(*xyz);
	}

	void PointRenderModule::display()
	{
		glMatrixMode(GL_MODELVIEW_MATRIX);
		glPushMatrix();

		glRotatef(m_rotation.x(), m_rotation.y(), m_rotation.z(), m_rotation.w());
		glTranslatef(m_translation[0], m_translation[1], m_translation[2]);
		glScalef(m_scale[0], m_scale[1], m_scale[2]);

 		//m_pointRender->display();

		switch (this->varRenderMode()->getReference()->currentKey())
		{
		case RenderModeEnum::POINT:
			m_pointRender->renderPoints();
			break;
		case RenderModeEnum::SPRITE:
			m_pointRender->renderSprite();
			break;
		case RenderModeEnum::INSTANCE:
			m_pointRender->renderInstancedSphere();
		default:
			break;
		}

		glPopMatrix();
	}

	void PointRenderModule::setColor(Vector3f color)
	{
		m_color = color;
	}

	void PointRenderModule::setColorRange(float min, float max)
	{
		m_minIndex.setValue(min);
		m_maxIndex.setValue(max);
	}

	void PointRenderModule::setReferenceColor(float v)
	{
		m_refV = v;
	}

}