#include "hip/hip_runtime.h"
#pragma once
#include <glad/gl.h>
#include "cuda_helper_math.h"
#include "SurfaceMeshRender.h"
#include "Topology/TriangleSet.h"
#include "Vector.h"
#include "Framework/Node.h"
#include "OpenGLContext.h"

namespace dyno
{
	IMPLEMENT_CLASS(SurfaceMeshRender)

		SurfaceMeshRender::SurfaceMeshRender()
		: VisualModule()
		, m_color(Vec3f(0.2f, 0.3, 0.0f))
	{
	}

	SurfaceMeshRender::~SurfaceMeshRender()
	{
		vertices.clear();
		normals.clear();
		colors.clear();
	}

	bool SurfaceMeshRender::initializeImpl()
	{
		m_triangleRender = std::make_shared<TriangleRender>();

		return true;
	}

	__global__ void SetupTriangles(
		DArray<float3> originVerts,
		DArray<float3> vertices,
		DArray<float3> normals,
		DArray<float3> colors,
		DArray<TopologyModule::Triangle> triangles,
		float3 color
		)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= triangles.size()) return;

		TopologyModule::Triangle tri = triangles[pId];
		float3 v1 = originVerts[tri[0]];
		float3 v2 = originVerts[tri[1]];
		float3 v3 = originVerts[tri[2]];

		vertices[3 * pId + 0] = v1;
		vertices[3 * pId + 1] = v2;
		vertices[3 * pId + 2] = v3;

		float3 triN = cross(v2-v1, v3-v1);
		triN = normalize(triN);

		normals[3 * pId + 0] = triN;
		normals[3 * pId + 1] = triN;
		normals[3 * pId + 2] = triN;

		colors[3 * pId + 0] = color;
		colors[3 * pId + 1] = color;
		colors[3 * pId + 2] = color;
	}

	void SurfaceMeshRender::updateRenderingContext()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return;
		}

		auto triSet = TypeInfo::cast<TriangleSet<DataType3f>>(parent->getTopologyModule());
		if (triSet == nullptr)
		{
			Log::sendMessage(Log::Error, "TriangleModule: The topology module is not supported!");
			return;
		}

		auto verts = triSet->getPoints();
		auto triangles = triSet->getTriangles();

		if (m_triangleRender->numberOfTrianlges() != triangles->size())
		{
			m_triangleRender->resize(triangles->size());
		
			normals.resize(3 * triangles->size());
			vertices.resize(3 * triangles->size());
			colors.resize(3 * triangles->size());
			
		}

		uint pDims = cudaGridSize(triangles->size(), BLOCK_SIZE);

		DArray<float3>* fverts = (DArray<float3>*)&verts;
		SetupTriangles << <pDims, BLOCK_SIZE >> >(*fverts, vertices, normals, colors, *triangles, make_float3(m_color[0], m_color[1], m_color[2]));
		cuSynchronize();

		m_triangleRender->setVertexArray(vertices);
		m_triangleRender->setColorArray(colors);
		m_triangleRender->setNormalArray(normals);
	}

	void SurfaceMeshRender::display()
	{
		//return;
		glMatrixMode(GL_MODELVIEW_MATRIX);
		glPushMatrix();

		glRotatef(m_rotation.x(), m_rotation.y(), m_rotation.z(), m_rotation.w());
		glTranslatef(m_translation[0], m_translation[1], m_translation[2]);
		glScalef(m_scale[0], m_scale[1], m_scale[2]);

		m_triangleRender->display();

		glPopMatrix();
	}

	void SurfaceMeshRender::setColor(Vec3f color)
	{
		m_color = color;
	}

}