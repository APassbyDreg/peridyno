#include "hip/hip_runtime.h"
#include "PointInteraction.h"
#include <thrust/sort.h>
#include <iostream>
#include <OrbitCamera.h>

namespace dyno
{
	__global__ void PointInitializeArray(
		DArray<int> intersected)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= intersected.size()) return;

		intersected[pId] = 0;
	}

	__global__ void PointMergeIntersectedIndex(
		DArray<int> intersected1,
		DArray<int> intersected2,
		DArray<int> outIntersected,
		DArray<int> outUnintersected)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= intersected1.size()) return;

		if (intersected1[pId] == 0 && intersected2[pId] == 0)
			outIntersected[pId] = 0;
		else
			outIntersected[pId] = 1;

		outUnintersected[pId] = outIntersected[pId] == 1 ? 0 : 1;
	}

	template<typename TDataType>
	PointInteraction<TDataType>::PointInteraction()
	{
		this->ray1 = TRay3D<Real>();
		this->ray2 = TRay3D<Real>();
		this->isPressed = false;
	}

	template<typename TDataType>
	void PointInteraction<TDataType>::onEvent(PMouseEvent event)
	{
		if (!event.altKeyPressed()) {
			if (camera == nullptr)
			{
				this->camera = event.camera;
			}
			this->varToggleMultiSelect()->setValue(false);
			if (event.controlKeyPressed()) 
			{
				this->varToggleMultiSelect()->setValue(true);
			}
			if (event.actionType == AT_PRESS)
			{
				this->camera = event.camera;
				this->isPressed = true;
				printf("Mouse pressed: Origin: %f %f %f; Direction: %f %f %f \n", event.ray.origin.x, event.ray.origin.y, event.ray.origin.z, event.ray.direction.x, event.ray.direction.y, event.ray.direction.z);
				this->ray1.origin = event.ray.origin;
				this->ray1.direction = event.ray.direction;
				this->x1 = event.x;
				this->y1 = event.y;
				this->calcIntersectClick();
			}
			else if (event.actionType == AT_RELEASE)
			{
				this->isPressed = false;
				printf("Mouse released: Origin: %f %f %f; Direction: %f %f %f \n", event.ray.origin.x, event.ray.origin.y, event.ray.origin.z, event.ray.direction.x, event.ray.direction.y, event.ray.direction.z);
				this->ray2.origin = event.ray.origin;
				this->ray2.direction = event.ray.direction;
				this->x2 = event.x;
				this->y2 = event.y;
			}
			else
			{
				printf("%f %f \n", event.x, event.y);
				printf("Mouse repeated: Origin: %f %f %f; Direction: %f %f %f \n", event.ray.origin.x, event.ray.origin.y, event.ray.origin.z, event.ray.direction.x, event.ray.direction.y, event.ray.direction.z);
				if (this->isPressed) {
					this->ray2.origin = event.ray.origin;
					this->ray2.direction = event.ray.direction;
					this->x2 = event.x;
					this->y2 = event.y;
					this->calcIntersectDrag();
				}
			}
		}
	}

	template <typename Real, typename Coord>
	__global__ void CalIntersectedPointsRay(
		DArray<Coord> points,
		DArray<int> intersected,
		DArray<int> unintersected,
		TRay3D<Real> mouseray,
		Real radius)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= points.size()) return;

		TSphere3D<Real> sphere = TSphere3D<Real>(points[pId], radius);
		TSegment3D<Real> seg;
		int temp = mouseray.intersect(sphere, seg);
		if (temp > 0 || intersected[pId] == 1)
			intersected[pId] = 1;
		else
			intersected[pId] = 0;
		unintersected[pId] = (intersected[pId] == 1 ? 0 : 1);
	}

	template <typename Real, typename Coord>
	__global__ void CalIntersectedPointsBox(
		DArray<Coord> points,
		DArray<int> intersected,
		DArray<int> unintersected,
		TPlane3D<Real> plane13,
		TPlane3D<Real> plane42,
		TPlane3D<Real> plane14,
		TPlane3D<Real> plane32)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= points.size()) return;

		bool flag = false;
		float temp1 = ((points[pId] - plane13.origin).dot(plane13.normal)) * ((points[pId] - plane42.origin).dot(plane42.normal));
		float temp2 = ((points[pId] - plane14.origin).dot(plane14.normal)) * ((points[pId] - plane32.origin).dot(plane32.normal));
		if (temp1 >= 0 && temp2 >= 0)
			flag = true;

		if (flag || intersected[pId] == 1)
			intersected[pId] = 1;
		else
			intersected[pId] = 0;
		unintersected[pId] = (intersected[pId] == 1 ? 0 : 1);
	}

	template <typename Coord>
	__global__ void AssignOutPoints(
		DArray<Coord> points,
		DArray<Coord> intersected_points,
		DArray<Coord> unintersected_points,
		DArray<int> intersected,
		DArray<int> unintersected,
		DArray<int> intersected_o)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= points.size()) return;

		if (intersected_o[pId] == 1)
		{
			intersected_points[intersected[pId]] = points[pId];
		}
		else
		{
			unintersected_points[unintersected[pId]] = points[pId];

		}
	}

	template<typename TDataType>
	void PointInteraction<TDataType>::calcPointIntersectClick()
	{
		TriangleSet<TDataType> initialTriangleSet = this->inInitialTriangleSet()->getData();
		DArray<Coord> points = initialTriangleSet.getPoints();
		DArray<int> intersected;
		intersected.resize(points.size());
		cuExecute(points.size(),
			PointInitializeArray,
			intersected
		);
		DArray<int> unintersected;
		unintersected.resize(points.size());
		std::cout << "Point Num:" << points.size() << std::endl;
		cuExecute(points.size(),
			CalIntersectedPointsRay,
			points,
			intersected,
			unintersected,
			this->ray1,
			this->varInterationRadius()->getData()
		);

		if (this->varToggleMultiSelect()->getData())
		{
			if (this->pointIntersectedIndex.size() == 0)
			{
				this->pointIntersectedIndex.resize(points.size());
				cuExecute(points.size(),
					PointInitializeArray,
					this->pointIntersectedIndex
				)
			}
			DArray<int> outIntersected;
			outIntersected.resize(intersected.size());
			DArray<int> outUnintersected;
			outUnintersected.resize(unintersected.size());
			cuExecute(points.size(),
				PointMergeIntersectedIndex,
				this->pointIntersectedIndex,
				intersected,
				outIntersected,
				outUnintersected
			);
			intersected.assign(outIntersected);
			unintersected.assign(outUnintersected);
			this->pointIntersectedIndex.assign(intersected);
		}
		else
		{
			this->pointIntersectedIndex.assign(intersected);
		}
		DArray<int> intersected_o;
		intersected_o.assign(intersected);

		int intersected_size = thrust::reduce(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), intersected.begin());
		DArray<Coord> intersected_points;
		intersected_points.resize(intersected_size);
		std::cout << intersected_size << std::endl;

		int unintersected_size = thrust::reduce(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), unintersected.begin());
		DArray<Coord> unintersected_points;
		unintersected_points.resize(unintersected_size);

		cuExecute(points.size(),
			AssignOutPoints,
			points,
			intersected_points,
			unintersected_points,
			intersected,
			unintersected,
			intersected_o
		);
		std::cout << "Selected Points Num:" << intersected_points.size() << std::endl;
		this->outSelectedPointSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outSelectedPointSet()->getDataPtr()->setPoints(intersected_points);
		this->outOtherPointSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outOtherPointSet()->getDataPtr()->setPoints(unintersected_points);
		this->outPointIndex()->getDataPtr()->assign(intersected_o);
	}

	template<typename TDataType>
	void PointInteraction<TDataType>::calcPointIntersectDrag()
	{
		TRay3D<Real> ray1 = this->ray1;
		TRay3D<Real> ray2 = this->ray2;
		TRay3D<Real> ray3 = this->camera->castRayInWorldSpace((float)x1, (float)y2);
		TRay3D<Real> ray4 = this->camera->castRayInWorldSpace((float)x2, (float)y1);

		TPlane3D<Real> plane13 = TPlane3D<Real>(ray1.origin, ray1.direction.cross(ray3.direction));
		TPlane3D<Real> plane42 = TPlane3D<Real>(ray2.origin, ray2.direction.cross(ray4.direction));
		TPlane3D<Real> plane14 = TPlane3D<Real>(ray4.origin, ray1.direction.cross(ray4.direction));
		TPlane3D<Real> plane32 = TPlane3D<Real>(ray3.origin, ray2.direction.cross(ray3.direction));

		TriangleSet<TDataType> initialTriangleSet = this->inInitialTriangleSet()->getData();
		DArray<Coord> points = initialTriangleSet.getPoints();
		DArray<int> intersected;
		intersected.resize(points.size());
		cuExecute(points.size(),
			PointInitializeArray,
			intersected
		);
		DArray<int> unintersected;
		unintersected.resize(points.size());
		std::cout << "Point Num:" << points.size() << std::endl;
		cuExecute(points.size(),
			CalIntersectedPointsBox,
			points,
			intersected,
			unintersected,
			plane13,
			plane42,
			plane14,
			plane32
		);
		cuExecute(points.size(),
			CalIntersectedPointsRay,
			points,
			intersected,
			unintersected,
			this->ray1,
			this->varInterationRadius()->getData()
		);

		if (this->varToggleMultiSelect()->getData())
		{
			if (this->pointIntersectedIndex.size() == 0)
			{
				this->pointIntersectedIndex.resize(points.size());
				cuExecute(points.size(),
					PointInitializeArray,
					this->pointIntersectedIndex
				)
			}
			DArray<int> outIntersected;
			outIntersected.resize(intersected.size());
			DArray<int> outUnintersected;
			outUnintersected.resize(unintersected.size());
			cuExecute(points.size(),
				PointMergeIntersectedIndex,
				this->pointIntersectedIndex,
				intersected,
				outIntersected,
				outUnintersected
			);
			intersected.assign(outIntersected);
			unintersected.assign(outUnintersected);
			this->pointIntersectedIndex.assign(intersected);
		}
		else
		{
			this->pointIntersectedIndex.assign(intersected);
		}

		DArray<int> intersected_o;
		intersected_o.assign(intersected);

		int intersected_size = thrust::reduce(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), intersected.begin());
		DArray<Coord> intersected_points;
		intersected_points.resize(intersected_size);

		int unintersected_size = thrust::reduce(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), unintersected.begin());
		DArray<Coord> unintersected_points;
		unintersected_points.resize(unintersected_size);

		cuExecute(points.size(),
			AssignOutPoints,
			points,
			intersected_points,
			unintersected_points,
			intersected,
			unintersected,
			intersected_o
		);
		std::cout << "Selected Points Num:" << intersected_points.size() << std::endl;
		this->outSelectedPointSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outSelectedPointSet()->getDataPtr()->setPoints(intersected_points);
		this->outOtherPointSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outOtherPointSet()->getDataPtr()->setPoints(unintersected_points);
		this->outPointIndex()->getDataPtr()->assign(intersected_o);
	}

	template<typename TDataType>
	void PointInteraction<TDataType>::calcIntersectClick()
	{
		if (this->varTogglePicker()->getData())
			calcPointIntersectClick();
	}

	template<typename TDataType>
	void PointInteraction<TDataType>::calcIntersectDrag()
	{
		if (this->varTogglePicker()->getData())
			calcPointIntersectDrag();
	}

	DEFINE_CLASS(PointInteraction);
}