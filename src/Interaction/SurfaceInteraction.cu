#include "hip/hip_runtime.h"
#include "SurfaceInteraction.h"
#include <thrust/sort.h>
#include <iostream>
#include <OrbitCamera.h>

namespace dyno
{
	__global__ void SurfaceInitializeArray(
		DArray<int> intersected)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= intersected.size()) return;

		intersected[pId] = 0;
	}

	__global__ void SurfaceMergeIntersectedIndex(
		DArray<int> intersected1,
		DArray<int> intersected2,
		DArray<int> outIntersected,
		DArray<int> outUnintersected)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= intersected1.size()) return;

		if (intersected1[pId] == 0 && intersected2[pId] == 0)
			outIntersected[pId] = 0;
		else
			outIntersected[pId] = 1;

		outUnintersected[pId] = outIntersected[pId] == 1 ? 0 : 1;
	}

	template<typename TDataType>
	SurfaceInteraction<TDataType>::SurfaceInteraction()
	{
		this->ray1 = TRay3D<Real>();
		this->ray2 = TRay3D<Real>();
		this->isPressed = false;
	}

	template<typename TDataType>
	void SurfaceInteraction<TDataType>::onEvent(PMouseEvent event)
	{
		if (!event.altKeyPressed()) {
			if (camera == nullptr)
			{
				this->camera = event.camera;
			}
			this->varToggleMultiSelect()->setValue(false);
			if (event.controlKeyPressed()) 
			{
				this->varToggleMultiSelect()->setValue(true);
			}
			if (event.actionType == AT_PRESS)
			{
				this->camera = event.camera;
				this->isPressed = true;
				printf("Mouse pressed: Origin: %f %f %f; Direction: %f %f %f \n", event.ray.origin.x, event.ray.origin.y, event.ray.origin.z, event.ray.direction.x, event.ray.direction.y, event.ray.direction.z);
				this->ray1.origin = event.ray.origin;
				this->ray1.direction = event.ray.direction;
				this->x1 = event.x;
				this->y1 = event.y;
				this->calcIntersectClick();
			}
			else if (event.actionType == AT_RELEASE)
			{
				this->isPressed = false;
				printf("Mouse released: Origin: %f %f %f; Direction: %f %f %f \n", event.ray.origin.x, event.ray.origin.y, event.ray.origin.z, event.ray.direction.x, event.ray.direction.y, event.ray.direction.z);
				this->ray2.origin = event.ray.origin;
				this->ray2.direction = event.ray.direction;
				this->x2 = event.x;
				this->y2 = event.y;
			}
			else
			{
				printf("%f %f \n", event.x, event.y);
				printf("Mouse repeated: Origin: %f %f %f; Direction: %f %f %f \n", event.ray.origin.x, event.ray.origin.y, event.ray.origin.z, event.ray.direction.x, event.ray.direction.y, event.ray.direction.z);
				if (this->isPressed) {
					this->ray2.origin = event.ray.origin;
					this->ray2.direction = event.ray.direction;
					this->x2 = event.x;
					this->y2 = event.y;
					this->calcIntersectDrag();
				}
			}
		}
	}

	template <typename Triangle, typename Real, typename Coord>
	__global__ void CalIntersectedTrisRay(
		DArray<Coord> points,
		DArray<Triangle> triangles,
		DArray<int> intersected,
		DArray<int> unintersected,
		TRay3D<Real> mouseray)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= triangles.size()) return;

		TTriangle3D<Real> t = TTriangle3D<Real>(points[triangles[pId].data[0]], points[triangles[pId].data[1]], points[triangles[pId].data[2]]);
		int temp = 0;
		if (mouseray.direction.dot(t.normal()) < 0)
		{
			TPoint3D<Real> p;
			temp = mouseray.intersect(t, p);
		}
		if (temp == 1 || intersected[pId] == 1)
			intersected[pId] = 1;
		else
			intersected[pId] = 0;
		unintersected[pId] = (intersected[pId] == 1 ? 0 : 1);
	}

	template <typename Triangle, typename Real, typename Coord>
	__global__ void CalIntersectedTrisBox(
		DArray<Coord> points,
		DArray<Triangle> triangles,
		DArray<int> intersected,
		DArray<int> unintersected,
		TPlane3D<Real> plane13,
		TPlane3D<Real> plane42,
		TPlane3D<Real> plane14,
		TPlane3D<Real> plane32,
		TRay3D<Real> mouseray)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= triangles.size()) return;


		TTriangle3D<Real> t = TTriangle3D<Real>(points[triangles[pId].data[0]], points[triangles[pId].data[1]], points[triangles[pId].data[2]]);
		bool flag = false;
		if (mouseray.direction.dot(t.normal()) < 0)
		{
			for (int i = 0; i < 3; i++) {
				float temp1 = ((points[triangles[pId].data[i]] - plane13.origin).dot(plane13.normal)) * ((points[triangles[pId].data[i]] - plane42.origin).dot(plane42.normal));
				float temp2 = ((points[triangles[pId].data[i]] - plane14.origin).dot(plane14.normal)) * ((points[triangles[pId].data[i]] - plane32.origin).dot(plane32.normal));
				if (temp1 >= 0 && temp2 >= 0)
				{
					flag = true;
					break;
				}
			}
		}
		if (flag || intersected[pId] == 1)
			intersected[pId] = 1;
		else
			intersected[pId] = 0;
		unintersected[pId] = (intersected[pId] == 1 ? 0 : 1);
	}

	template <typename Triangle>
	__global__ void AssignOutTriangles(
		DArray<Triangle> triangles,
		DArray<Triangle> intersected_triangles,
		DArray<Triangle> unintersected_triangles,
		DArray<int> intersected,
		DArray<int> unintersected,
		DArray<int> intersected_o)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= triangles.size()) return;

		if (intersected_o[pId] == 1)
		{
			intersected_triangles[intersected[pId]] = triangles[pId];
		}
		else
		{
			unintersected_triangles[unintersected[pId]] = triangles[pId];

		}
	}

	template<typename TDataType>
	void SurfaceInteraction<TDataType>::calcSurfaceIntersectClick()
	{
		TriangleSet<TDataType> initialTriangleSet = this->inInitialTriangleSet()->getData();
		DArray<Coord> points = initialTriangleSet.getPoints();
		DArray<Triangle> triangles = initialTriangleSet.getTriangles();
		DArray<int> intersected;
		intersected.resize(triangles.size());
		cuExecute(triangles.size(),
			SurfaceInitializeArray,
			intersected
		);
		DArray<int> unintersected;
		unintersected.resize(triangles.size());
		std::cout << "Triangle Num:" << triangles.size() << std::endl;
		cuExecute(triangles.size(),
			CalIntersectedTrisRay,
			points,
			triangles,
			intersected,
			unintersected,
			this->ray1
		);

		if (this->varToggleMultiSelect()->getData())
		{
			if (this->triIntersectedIndex.size() == 0) 
			{
				this->triIntersectedIndex.resize(triangles.size());
				cuExecute(triangles.size(),
					SurfaceInitializeArray,
					this->triIntersectedIndex
				)
			}
			DArray<int> outIntersected;
			outIntersected.resize(intersected.size());
			DArray<int> outUnintersected;
			outUnintersected.resize(unintersected.size());
			cuExecute(triangles.size(),
				SurfaceMergeIntersectedIndex,
				this->triIntersectedIndex,
				intersected,
				outIntersected,
				outUnintersected
			);
			intersected.assign(outIntersected);
			unintersected.assign(outUnintersected);
			this->triIntersectedIndex.assign(intersected);
		}
		else
		{
			this->triIntersectedIndex.assign(intersected);
		}

		DArray<int> intersected_o;
		intersected_o.assign(intersected);

		int intersected_size = thrust::reduce(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), intersected.begin());
		DArray<Triangle> intersected_triangles;
		intersected_triangles.resize(intersected_size);

		int unintersected_size = thrust::reduce(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), unintersected.begin());
		DArray<Triangle> unintersected_triangles;
		unintersected_triangles.resize(unintersected_size);

		cuExecute(triangles.size(),
			AssignOutTriangles,
			triangles,
			intersected_triangles,
			unintersected_triangles,
			intersected,
			unintersected,
			intersected_o
		);
		std::cout << "Selected Triangles Num:" << intersected_triangles.size() << std::endl;
		this->outSelectedTriangleSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outSelectedTriangleSet()->getDataPtr()->setTriangles(intersected_triangles);
		this->outOtherTriangleSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outOtherTriangleSet()->getDataPtr()->setTriangles(unintersected_triangles);
		this->outTriangleIndex()->getDataPtr()->assign(intersected_o);
	}

	template<typename TDataType>
	void SurfaceInteraction<TDataType>::calcSurfaceIntersectDrag()
	{
		TRay3D<Real> ray1 = this->ray1;
		TRay3D<Real> ray2 = this->ray2;
		TRay3D<Real> ray3 = this->camera->castRayInWorldSpace((float)x1, (float)y2);
		TRay3D<Real> ray4 = this->camera->castRayInWorldSpace((float)x2, (float)y1);

		TPlane3D<Real> plane13 = TPlane3D<Real>(ray1.origin, ray1.direction.cross(ray3.direction));
		TPlane3D<Real> plane42 = TPlane3D<Real>(ray2.origin, ray2.direction.cross(ray4.direction));
		TPlane3D<Real> plane14 = TPlane3D<Real>(ray4.origin, ray1.direction.cross(ray4.direction));
		TPlane3D<Real> plane32 = TPlane3D<Real>(ray3.origin, ray2.direction.cross(ray3.direction));

		TriangleSet<TDataType> initialTriangleSet = this->inInitialTriangleSet()->getData();
		DArray<Coord> points = initialTriangleSet.getPoints();
		DArray<Triangle> triangles = initialTriangleSet.getTriangles();
		DArray<int> intersected;
		intersected.resize(triangles.size());
		cuExecute(triangles.size(),
			SurfaceInitializeArray,
			intersected
		);
		DArray<int> unintersected;
		unintersected.resize(triangles.size());
		std::cout << "Triangle Num:" << triangles.size() << std::endl;
		cuExecute(triangles.size(),
			CalIntersectedTrisBox,
			points,
			triangles,
			intersected,
			unintersected,
			plane13,
			plane42,
			plane14,
			plane32,
			this->ray2
		);
		cuExecute(triangles.size(),
			CalIntersectedTrisRay,
			points,
			triangles,
			intersected,
			unintersected,
			this->ray1
		);

		if (this->varToggleMultiSelect()->getData())
		{
			if (this->triIntersectedIndex.size() == 0)
			{
				this->triIntersectedIndex.resize(triangles.size());
				cuExecute(triangles.size(),
					SurfaceInitializeArray,
					this->triIntersectedIndex
				)
			}
			DArray<int> outIntersected;
			outIntersected.resize(intersected.size());
			DArray<int> outUnintersected;
			outUnintersected.resize(unintersected.size());
			cuExecute(triangles.size(),
				SurfaceMergeIntersectedIndex,
				this->triIntersectedIndex,
				intersected,
				outIntersected,
				outUnintersected
			);
			intersected.assign(outIntersected);
			unintersected.assign(outUnintersected);
			this->triIntersectedIndex.assign(intersected);
		}
		else
		{
			this->triIntersectedIndex.assign(intersected);
		}


		DArray<int> intersected_o;
		intersected_o.assign(intersected);

		int intersected_size = thrust::reduce(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, intersected.begin(), intersected.begin() + intersected.size(), intersected.begin());
		DArray<Triangle> intersected_triangles;
		intersected_triangles.resize(intersected_size);

		int unintersected_size = thrust::reduce(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, unintersected.begin(), unintersected.begin() + unintersected.size(), unintersected.begin());
		DArray<Triangle> unintersected_triangles;
		unintersected_triangles.resize(unintersected_size);

		cuExecute(triangles.size(),
			AssignOutTriangles,
			triangles,
			intersected_triangles,
			unintersected_triangles,
			intersected,
			unintersected,
			intersected_o
		);
		std::cout << "Selected Triangles Num:" << intersected_triangles.size() << std::endl;
		this->outSelectedTriangleSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outSelectedTriangleSet()->getDataPtr()->setTriangles(intersected_triangles);
		this->outOtherTriangleSet()->getDataPtr()->copyFrom(initialTriangleSet);
		this->outOtherTriangleSet()->getDataPtr()->setTriangles(unintersected_triangles);
		this->outTriangleIndex()->getDataPtr()->assign(intersected_o);
	}

	template<typename TDataType>
	void SurfaceInteraction<TDataType>::calcIntersectClick()
	{
		if (this->varTogglePicker()->getData())
			calcSurfaceIntersectClick();
	}

	template<typename TDataType>
	void SurfaceInteraction<TDataType>::calcIntersectDrag()
	{
		if (this->varTogglePicker()->getData())
			calcSurfaceIntersectDrag();
	}

	DEFINE_CLASS(SurfaceInteraction);
}