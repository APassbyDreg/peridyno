#include "hip/hip_runtime.h"
#include "NeighborElementQuery.h"
#include "CollisionDetectionAlgorithm.h"

#include "Collision/CollisionDetectionBroadPhase.h"

#include "Topology/Primitive3D.h"

namespace dyno
{
	IMPLEMENT_CLASS_1(NeighborElementQuery, TDataType)
		typedef typename TOrientedBox3D<Real> Box3D;


	template<typename TDataType>
	NeighborElementQuery<TDataType>::NeighborElementQuery()
		: ComputeModule()
	{
		this->inRadius()->setValue(Real(0.011));

		m_broadPhaseCD = std::make_shared<CollisionDetectionBroadPhase<TDataType>>();
		//fout.open("data_Oct_without_arrange.txt");
	}

	template<typename TDataType>
	NeighborElementQuery<TDataType>::~NeighborElementQuery()
	{
	}

	template<typename Real, typename Coord>
	__global__ void NEQ_SetupAABB(
		DArray<AABB> boundingBox,
		DArray<Coord> position,
		Real radius)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		AABB box;
		Coord p = position[pId];
		box.v0 = p - radius;
		box.v1 = p + radius;

		boundingBox[pId] = box;
	}

	template<typename Box3D>
	__global__ void NEQ_SetupAABB(
		DArray<AABB> boundingBox,
		DArray<Box3D> boxes,
		DArray<Sphere3D> spheres,
		DArray<Tet3D> tets,
		DArray<Capsule3D> caps,
		DArray<Triangle3D> tris,
		ElementOffset elementOffset,
		Real boundary_expand)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		ElementType eleType = checkElementType(tId, elementOffset);

		//Real boundary_expand = 0.0075f;

		AABB box;
		switch (eleType)
		{
		case CT_SPHERE:
		{
			box = spheres[tId].aabb();

			break;
		}
		case CT_BOX:
		{
			box = boxes[tId - elementOffset.boxOffset].aabb();
			box.v0 -= boundary_expand;
			box.v1 += boundary_expand;
			break;
		}
		case CT_TET:
		{
			box = tets[tId - elementOffset.tetOffset].aabb();
			box.v0 -= boundary_expand;
			box.v1 += boundary_expand;
			break;
		}
		case CT_SEG:
		{
			box = caps[tId - elementOffset.segOffset].aabb();
			box.v0 -= boundary_expand;
			box.v1 += boundary_expand;
			break;
		}
		case CT_TRI:
		{
			box = tris[tId - elementOffset.triOffset].aabb();
			box.v0 -= boundary_expand;
			box.v1 += boundary_expand;
			box.v0 -= boundary_expand;
			box.v1 += boundary_expand;
			/*printf("%.3lf %.3lf %.3lf\n%.3lf %.3lf %.3lf\n=========\n",
				box.v0[0], box.v0[1], box.v0[2],
				box.v1[0], box.v1[1], box.v1[2]);*/
			break;
		}
		default:
			break;
		}

		boundingBox[tId] = box;
	}

	


	template<typename Box3D>
	__global__ void NEQ_Narrow_Count(
		DArrayList<int> nbr,
		DArray<Box3D> boxes,
		DArray<Sphere3D> spheres,
		DArray<Tet3D> tets,
		DArray<Real> tets_sdf,
		DArray<int> tet_body_ids,
		DArray<TopologyModule::Tetrahedron> tet_element_ids,
		DArray<Capsule3D> caps,
		DArray<Triangle3D> triangles,
		DArray<int> count,
		ElementOffset elementOffset,
		NbrFilter filter,
		Real boundary_expand)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= nbr.size()) return;

		ElementType eleType_i = checkElementType(tId, elementOffset);

		int start_box = elementOffset.boxOffset;
		int start_tet = elementOffset.tetOffset;
		//Real boundary_expand = 0.0;
		int cnt = 0;

		switch (eleType_i)
		{
		case CT_SPHERE:
		{
			//break;
			//int nbSize = nbr.getNeighborSize(tId);
			//if(nbSize != 0)
				//printf("nbSize = %d\n", nbSize);
			List<int>& list_i = nbr[tId];
			int nbSize = list_i.size();

			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);

				//printf("%d %d\n", eleType_i, eleType_j);

				switch (eleType_j)
				{
				case CT_SPHERE:
				{

					Real proj_dist = (spheres[j].center - spheres[tId].center).norm();
					if (filter.sphere_sphere && proj_dist < spheres[tId].radius + spheres[j].radius)
					{
						cnt++;
					}
					break;
				}
				case CT_BOX:
				{
					Point3D pos_sphere(spheres[tId].center);
					Coord3D proj_pos = pos_sphere.project(boxes[j - elementOffset.boxOffset]).origin;

					if (filter.sphere_box)
						if ((proj_pos - spheres[tId].center).norm() < spheres[tId].radius + boundary_expand
							|| pos_sphere.inside(boxes[j - elementOffset.boxOffset]))
						{
							cnt++;

						}
					break;
				}
				case CT_TET:
				{
					//printf("@@@@@@@@@@@@@@@@@@ %d\n", tId);
					if (filter.sphere_tet && filter.tet_sdf && tets_sdf.size() > 0)
					{
						Point3D pos_sphere(spheres[tId].center);
						int idx;
						Bool tmp;
						Coord3D proj_pos = pos_sphere.project(tets[j - elementOffset.tetOffset], tmp, &idx).origin;

						int id1 = (idx + 1) % 4;
						int id2 = (idx + 2) % 4;
						int id3 = (idx + 3) % 4;

						Triangle3D t3d_tmp = Triangle3D(
							tets[j - elementOffset.tetOffset].v[id1],
							tets[j - elementOffset.tetOffset].v[id2],
							tets[j - elementOffset.tetOffset].v[id3]
						);

						if (tets_sdf[4 * (j - elementOffset.tetOffset) + id1] < EPSILON &&
							tets_sdf[4 * (j - elementOffset.tetOffset) + id2] < EPSILON &&
							tets_sdf[4 * (j - elementOffset.tetOffset) + id3] < EPSILON)
						{
							if (((proj_pos - spheres[tId].center).norm() < spheres[tId].radius + boundary_expand
								&& (((proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm()).cross(t3d_tmp.normal() / t3d_tmp.normal().norm())).norm() < EPSILON)
								|| pos_sphere.inside(tets[j - elementOffset.tetOffset]))
							{
								cnt++;
							}
						}
						else if (pos_sphere.inside(tets[j - elementOffset.tetOffset]))
						{
							cnt++;
						}
					}
					else if (filter.sphere_tet)
					{
						Point3D pos_sphere(spheres[tId].center);
						int idx;
						Bool tmp;
						Coord3D proj_pos = pos_sphere.project(tets[j - elementOffset.tetOffset], tmp, &idx).origin;
						if ((proj_pos - spheres[tId].center).norm() < spheres[tId].radius + boundary_expand
							|| pos_sphere.inside(tets[j - elementOffset.tetOffset]))
						{
							cnt++;
						}
					}
					break;
				}
				case CT_SEG:
				{
					Point3D pos_sphere(spheres[tId].center);
					Segment3D segment_tmp = caps[j - elementOffset.segOffset].segment;
					if (filter.sphere_capsule)
						if (pos_sphere.distance(segment_tmp) < spheres[tId].radius + caps[j - elementOffset.segOffset].radius + boundary_expand)
						{
							//printf("CNT_OKKKK\n");
							//if((j - elementOffset.segOffset) % 39 == 0 || (j - elementOffset.segOffset) % 39 == 38)
							cnt++;
							/*else if ((pos_sphere.project(segment_tmp).origin - pos_sphere.origin).dot(segment_tmp.direction()) < EPSILON)
							{
								cnt++;
							}*/
						}
					break;
				}
				case CT_TRI:
				{
					Point3D pos_sphere(spheres[tId].center);
					Triangle3D tri_tmp = triangles[j - elementOffset.triOffset];
					if (filter.sphere_tri)
						if (pos_sphere.distance(tri_tmp) < spheres[tId].radius + 1.75 * boundary_expand && pos_sphere.distance(tri_tmp) > EPSILON
							&& (((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).cross(tri_tmp.normal() / tri_tmp.normal().norm())).norm() < 0.001
							)
						{
							//printf("CNT_OKKKK\n");
							cnt++;
						}
					break;
				}
				default:
					break;
				}
			}
			break;

		}
		case CT_BOX:
		{
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			int nbSize = list_i.size();

			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);

				switch (eleType_j)
				{
				case CT_SPHERE:
				{
					Point3D pos_sphere(spheres[j].center);
					Coord3D proj_pos = pos_sphere.project(boxes[tId - elementOffset.boxOffset]).origin;
					if (filter.sphere_box)
						if ((proj_pos - spheres[j].center).norm() < spheres[j].radius + boundary_expand
							|| pos_sphere.inside(boxes[tId - elementOffset.boxOffset]))
						{
							cnt++;
						}
					//printf("sphere!!!! %d\n", j);
					break;
				}
				case CT_BOX:
				{
					TManifold<Real> manifold;

					auto boxA = boxes[tId - elementOffset.boxOffset];
					auto boxB = boxes[j - elementOffset.boxOffset];
					CollisionDetection<Real>::request(manifold, boxA, boxB);

					cnt += manifold.contactCount;

// 					Coord3D inter_norm, p1, p2;
// 					Real inter_dist;
// 					if (filter.box_box)
// 					{
// 						if (boxes[tId - elementOffset.boxOffset].point_intersect(boxes[j - elementOffset.boxOffset], inter_norm, inter_dist, p1, p2))
// 						{
// 							cnt++;
// 						}
// 						else if (boxes[j - elementOffset.boxOffset].point_intersect(boxes[tId - elementOffset.boxOffset], inter_norm, inter_dist, p1, p2))
// 						{
// 							cnt++;
// 						}
// 					}
					break;
				}
				case CT_TET:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.box_tet)
						if (boxes[tId - elementOffset.boxOffset].point_intersect(tets[j - elementOffset.tetOffset], inter_norm, inter_dist, p1, p2))
						{
							cnt++;
						}
					break;
				}
				case CT_SEG:
				{
					if (filter.box_capsule)
					{
						Segment3D segment_tmp = caps[j - elementOffset.segOffset].segment;
						Segment3D segment_intersect;
						Segment3D seg_prox = segment_tmp.proximity(boxes[tId - elementOffset.boxOffset]);
						if (segment_tmp.intersect(boxes[tId - elementOffset.boxOffset], segment_intersect))
						{
							cnt++;
						}
						else if (seg_prox.length() < caps[j - elementOffset.segOffset].radius)
						{
							//if(seg_prox.direction().dot(segment_tmp.direction()) < EPSILON)//////to delete
							cnt++;
						}
					}
					break;
				}
				case CT_TRI:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.box_tri)
					{
						if (boxes[tId - elementOffset.boxOffset].point_intersect(triangles[j - elementOffset.triOffset], inter_norm, inter_dist, p1, p2))
						{
							cnt++;
						}
					}
					break;
				}
				default:
					break;
				}
			}
			break;
		}
		case CT_TET:
		{
			//printf("nbSize = %d   %d\n", tId, nbr.getNeighborSize(tId));

			//int nbSize = nbr.getNeighborSize(tId);
			//if(nbSize != 0)
			List<int>& list_i = nbr[tId];
			int nbSize = list_i.size();
			for (int ne = 0; ne < nbSize; ne++)
			{
				//if(nbr.getElementSize() <= nbr.getElementIndex(tId, ne))
				//	printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);
				
				//break;
				//if(false)
				switch (eleType_j)
				{
				case CT_SPHERE:
				{
					//Point3D pos_sphere(spheres[j].center);
					//Coord3D proj_pos = pos_sphere.project(tets[tId - elementOffset.tetOffset]).origin;
					if (filter.sphere_tet)
					{
						//printf("j idx = %d   %d\n", j, spheres.size());
						Point3D pos_sphere(spheres[j].center);
						int idx;
						Bool tmp;

						Coord3D proj_pos = pos_sphere.project(tets[tId - elementOffset.tetOffset], tmp, &idx).origin;

						if (filter.tet_sdf && tets_sdf.size() > 0)
						{
							int id1 = (idx + 1) % 4;
							int id2 = (idx + 2) % 4;
							int id3 = (idx + 3) % 4;

							Triangle3D t3d_tmp = Triangle3D(
								tets[tId - elementOffset.tetOffset].v[id1],
								tets[tId - elementOffset.tetOffset].v[id2],
								tets[tId - elementOffset.tetOffset].v[id3]
							);

							if (tets_sdf[4 * (tId - elementOffset.tetOffset) + id1] < EPSILON &&
								tets_sdf[4 * (tId - elementOffset.tetOffset) + id2] < EPSILON &&
								tets_sdf[4 * (tId - elementOffset.tetOffset) + id3] < EPSILON)
							{
								if (((proj_pos - spheres[j].center).norm() < spheres[j].radius + boundary_expand
									&& (((proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm()).cross(t3d_tmp.normal() / t3d_tmp.normal().norm())).norm() < EPSILON)
									|| pos_sphere.inside(tets[tId - elementOffset.tetOffset]))
								{
									cnt++;
								}
							}
							else
							{
								if (pos_sphere.inside(tets[tId - elementOffset.tetOffset]))
								{
									cnt++;
								}
							}
						}
						else
						{
							if ((proj_pos - spheres[j].center).norm() < spheres[j].radius + boundary_expand
								|| pos_sphere.inside(tets[tId - elementOffset.tetOffset]))
							{
								cnt++;
							}
						}
					}
					break;
				}
				case CT_BOX:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.box_tet)
						if (boxes[j - elementOffset.boxOffset].point_intersect(tets[tId - elementOffset.tetOffset], inter_norm, inter_dist, p1, p2))
						{
							cnt++;
						}
					break;
				}
				case CT_TET:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.tet_tet)
					{

						bool tmp_tet = false;
						AABB interBox;
						if (!tets[tId - elementOffset.tetOffset].aabb().intersect(tets[j - elementOffset.tetOffset].aabb(), interBox))
						{
							break;
						}
						if (filter.tet_neighbor_filter
							&& tet_body_ids.size() > 0
							&& tet_body_ids[tId - elementOffset.tetOffset] == tet_body_ids[j - elementOffset.tetOffset])
						{
							bool tmp = false;
							for (int iii = 0; iii < 4; iii++)
							{
								for (int jjj = 0; jjj < 4; jjj++)
									if (tet_element_ids[tId - elementOffset.tetOffset][iii]
										== tet_element_ids[j - elementOffset.tetOffset][jjj])
									{
										tmp = true;
										break;
									}
								if (tmp)
									break;
							}
							if (tmp)
								break;

						}
						if (tets[tId - elementOffset.tetOffset].intersect(tets[j - elementOffset.tetOffset], inter_norm, inter_dist, p1, p2, false))
						{
							if (abs(inter_dist) > 3 * EPSILON)
							{
								cnt++;
								tmp_tet = true;
							}
						}
						if (tets[j - elementOffset.tetOffset].intersect(tets[tId - elementOffset.tetOffset], inter_norm, inter_dist, p1, p2, false))
						{
							if(abs(inter_dist) > 3 * EPSILON && tmp_tet == false)
							{ 
								cnt++;
							}
						}
						
					}
					break;
				}
				case CT_SEG:
				{
					if (filter.tet_capsule)
					{
						Segment3D segment_tmp = caps[j - elementOffset.segOffset].segment;
						Line3D line_tmp(segment_tmp.v0, segment_tmp.direction());
						Segment3D segment_tmp2 = segment_tmp.proximity(tets[tId - elementOffset.tetOffset]);
						Segment3D segment_intersect;
						bool intersect_1 = false;
						if (line_tmp.intersect(tets[tId - elementOffset.tetOffset], segment_intersect))
						{
							Real left = (segment_intersect.v0 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							Real right = (segment_intersect.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < left)
							{
								Real tmp = left;
								left = right;
								right = tmp;
							}
							Real maxx = (segment_tmp.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < 0 || left > maxx)
							{

							}
							else
							{
								intersect_1 = true;
								cnt++;
							}
						}
						if (!intersect_1)
						{
							if (segment_tmp2.length() < caps[j - elementOffset.segOffset].radius)
							{
								cnt++;
							}
						}
					}
					break;
				}
				case CT_TRI:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.tet_tri)
					{
						if (tets[tId - elementOffset.tetOffset].intersect(triangles[j - elementOffset.triOffset], inter_norm, inter_dist, p1, p2))
						{
							cnt++;
						}
					}
					break;
				}
				default:
					break;
				}
			}
			break;
		}
		case CT_SEG:
		{
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			int nbSize = list_i.size();
			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);

				switch (eleType_j)
				{

				case CT_SPHERE:
				{
					Point3D pos_sphere(spheres[j].center);
					Segment3D segment_tmp = caps[tId - elementOffset.segOffset].segment;
					if (filter.sphere_capsule)
						if (pos_sphere.distance(segment_tmp) < spheres[j].radius + caps[tId - elementOffset.segOffset].radius + boundary_expand)
						{
							//printf("CNT_OKKKK\n");
							//cnt++;
							//printf("CNT_OKKKK\n");
							//if ((tId - elementOffset.segOffset) % 39 == 0 || (tId - elementOffset.segOffset) % 39 == 38)
							cnt++;
							/*else if ((pos_sphere.project(segment_tmp).origin - pos_sphere.origin).dot(segment_tmp.direction()) < EPSILON)
							{
								cnt++;
							}*/
						}
					break;
				}
				case CT_BOX:
				{
					if (filter.box_capsule)
					{
						Segment3D segment_tmp = caps[tId - elementOffset.segOffset].segment;
						Segment3D segment_intersect;
						Segment3D seg_prox = segment_tmp.proximity(boxes[j - elementOffset.boxOffset]);
						if (segment_tmp.intersect(boxes[j - elementOffset.boxOffset], segment_intersect))
						{
							cnt++;
						}
						else if (seg_prox.length() < caps[tId - elementOffset.segOffset].radius)
						{
							//if (seg_prox.direction().dot(segment_tmp.direction()) < EPSILON)//////to delete
							cnt++;
						}
					}
					break;
				}
				case CT_TET:
				{
					if (filter.tet_capsule)
					{
						Segment3D segment_tmp = caps[tId - elementOffset.segOffset].segment;
						Line3D line_tmp(segment_tmp.v0, segment_tmp.direction());
						Segment3D segment_tmp2 = segment_tmp.proximity(tets[j - elementOffset.tetOffset]);
						Segment3D segment_intersect;
						bool intersect_1 = false;
						if (line_tmp.intersect(tets[j - elementOffset.tetOffset], segment_intersect))
						{
							Real left = (segment_intersect.v0 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							Real right = (segment_intersect.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < left)
							{
								Real tmp = left;
								left = right;
								right = tmp;
							}
							Real maxx = (segment_tmp.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < 0 || left > maxx)
							{

							}
							else
							{
								intersect_1 = true;
								cnt++;
							}
						}
						if (!intersect_1)
						{
							if (segment_tmp2.length() < caps[tId - elementOffset.segOffset].radius)
							{
								cnt++;
							}
						}
					}
					break;
				}
				case CT_SEG:
				{
					if (filter.capsule_capsule)
					{
						Segment3D segment_1 = caps[j - elementOffset.segOffset].segment;
						Segment3D segment_2 = caps[tId - elementOffset.segOffset].segment;
						if ((segment_1.proximity(segment_2)).length() < caps[j - elementOffset.segOffset].radius + caps[tId - elementOffset.segOffset].radius)
						{
							if (abs(j - tId) >= 3)
								cnt++;
						}
					}
					break;
				}
				case CT_TRI:
				{
					if (filter.capsule_tri)
					{
						//Segment3D segment_1 = caps[j - elementOffset.segOffset].segment;
						Segment3D segment = caps[tId - elementOffset.segOffset].segment;
						Triangle3D triangle = triangles[j - elementOffset.triOffset];
						Point3D p3d;
						if (segment.intersect(triangle, p3d))
						{
							////if (abs(j - tId) >= 3)
							cnt++;
						}
						else if ((segment.proximity(triangle)).length() < caps[tId - elementOffset.segOffset].radius)
						{
							cnt++;
						}
					}
					break;
				}
				default:
					break;
				}
			}
			break;
		}
		case CT_TRI:
		{
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			int nbSize = list_i.size();

			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);

				switch (eleType_j)
				{

				case CT_SPHERE:
				{
					Point3D pos_sphere(spheres[j].center);
					Triangle3D tri_tmp = triangles[tId - elementOffset.triOffset];
					if (filter.sphere_tri)
						if (pos_sphere.distance(tri_tmp) < spheres[j].radius + 1.75 * boundary_expand && pos_sphere.distance(tri_tmp) > EPSILON
							//&& ((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).dot(tri_tmp.normal() / tri_tmp.normal().norm()) < EPSILON
							&& (((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).cross(tri_tmp.normal() / tri_tmp.normal().norm())).norm() < 0.001
							)
						{
							//printf("CNT_OKKKK\n");
							cnt++;
						}
					break;
				}
				case CT_BOX:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.box_tri)
					{
						if (boxes[j - elementOffset.boxOffset].point_intersect(triangles[tId - elementOffset.triOffset], inter_norm, inter_dist, p1, p2))
						{
							cnt++;
						}
					}
					break;
				}
				case CT_TET:
				{

					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.tet_tri)
					{
						if (tets[j - elementOffset.tetOffset].intersect(triangles[tId - elementOffset.triOffset], inter_norm, inter_dist, p1, p2))
						{
							cnt++;
						}
					}
					break;
				}
				case CT_SEG:
				{
					if (filter.capsule_tri)
					{
						//Segment3D segment_1 = caps[j - elementOffset.segOffset].segment;
						Segment3D segment = caps[j - elementOffset.segOffset].segment;
						Triangle3D triangle = triangles[tId - elementOffset.triOffset];
						Point3D p3d;
						if (segment.intersect(triangle, p3d))
						{
							cnt++;
						}
						else if ((segment.proximity(triangle)).length() < caps[j - elementOffset.segOffset].radius)
						{
							cnt++;
						}
					}
					break;
				}
				case CT_TRI:
				{

					break;
				}
				default:
					break;
				}
			}
			break;
		}
		default:
			break;
		}

		count[tId] = cnt;
	}

	template<typename Box3D, typename NeighborConstraints>
	__global__ void NEQ_Narrow_Set(
		DArrayList<int> nbr,
		DArray<Box3D> boxes,
		DArray<Sphere3D> spheres,
		DArray<Tet3D> tets,
		DArray<Real> tets_sdf,
		DArray<int> tet_body_ids,
		DArray<TopologyModule::Tetrahedron> tet_element_ids,
		DArray<Capsule3D> caps,
		DArray<Triangle3D> tris,
		DArray<NeighborConstraints> nbr_cons,
		DArray<int> prefix,
		ElementOffset elementOffset,
		NbrFilter filter,
		Real boundary_expand)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= nbr.size()) return;
		int cnt = 0;

		ElementType eleType_i = checkElementType(tId, elementOffset);
		//printf("box offset: %d\n", elementOffset.boxOffset);
		switch (eleType_i)
		{
		case CT_SPHERE:
		{
			//break;
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			int nbSize = list_i.size();
			for (int ne = 0; ne < nbSize; ne++)
			{
				int j = list_i[ne];
				ElementType eleType_j = checkElementType(j, elementOffset);
				switch (eleType_j)
				{
				case CT_SPHERE:
				{
					Real proj_dist = (spheres[j].center - spheres[tId].center).norm();
					if (filter.sphere_sphere)
						if (proj_dist < spheres[tId].radius + spheres[j].radius)
						{

							Coord3D inter_norm = (spheres[j].center - spheres[tId].center) / proj_dist;
							Coord3D p1 = spheres[j].center - inter_norm * spheres[j].radius;
							Coord3D p2 = spheres[tId].center + inter_norm * spheres[tId].radius;
							Real inter_dist = spheres[tId].radius + spheres[j].radius - proj_dist;

							//nbr_out.setElement(tId, cnt, j);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;//nbr_out.getElementIndex(tId, cnt);
							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = inter_dist;
							cnt++;
							//printf("?????????????\n");

						}
					break;
				}
				case CT_BOX:
				{
					if (filter.sphere_box)
					{
						Point3D pos_sphere(spheres[tId].center);
						Coord3D proj_pos = pos_sphere.project(boxes[j - elementOffset.boxOffset]).origin;
						if (pos_sphere.inside(boxes[j - elementOffset.boxOffset]))
						{
							Coord3D inter_norm = -(proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
							Coord3D p1 = proj_pos;
							Coord3D p2 = spheres[tId].center - inter_norm * spheres[tId].radius;
							Real inter_dist = spheres[tId].radius + (proj_pos - spheres[tId].center).norm();

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;//nbr_out.getElementIndex(tId, cnt);
							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							//nbr_cons[idx_con] = NeighborConstraints(j, tId, 8, p1, p2, 0.0f, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = inter_dist;
							cnt++;
						}
						else if ((proj_pos - spheres[tId].center).norm() < spheres[tId].radius + boundary_expand)
						{
							Coord3D inter_norm = (proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
							Coord3D p1 = proj_pos;
							Coord3D p2 = spheres[tId].center + inter_norm * spheres[tId].radius;
							Real inter_dist = spheres[tId].radius - (proj_pos - spheres[tId].center).norm();

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;//nbr_out.getElementIndex(tId, cnt);
							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_STICKINESS, p1, p2, inter_norm, Coord3D(0, 0, 0));
							//nbr_cons[idx_con] = NeighborConstraints(j, tId, 6, p1, p2, 0.0f, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = 0.0f;//inter_dist;
							cnt++;
							//printf("sphere!! %d\n", tId);
						}
					}
					break;
				}
				case CT_TET:
				{
					if (filter.sphere_tet)
					{
						Point3D pos_sphere(spheres[tId].center);
						int idx;
						Bool tmp;
						Coord3D proj_pos = pos_sphere.project(tets[j - elementOffset.tetOffset], tmp, &idx).origin;

						if (filter.tet_sdf)// && tets_sdf.size() > 0)
						{
							int id1 = (idx + 1) % 4;
							int id2 = (idx + 2) % 4;
							int id3 = (idx + 3) % 4;

							Triangle3D t3d_tmp = Triangle3D(
								tets[j - elementOffset.tetOffset].v[id1],
								tets[j - elementOffset.tetOffset].v[id2],
								tets[j - elementOffset.tetOffset].v[id3]
							);

							if (tets_sdf[4 * (j - elementOffset.tetOffset) + id1] < EPSILON &&
								tets_sdf[4 * (j - elementOffset.tetOffset) + id2] < EPSILON &&
								tets_sdf[4 * (j - elementOffset.tetOffset) + id3] < EPSILON)
							{
								//printf("========aaaa=aaaa======\n");
								if (pos_sphere.inside(tets[j - elementOffset.tetOffset]))
								{
									//printf("========bbbbb======\n");
									Coord3D inter_norm = -(proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
									Coord3D p1 = proj_pos;
									Coord3D p2 = spheres[tId].center - inter_norm * spheres[tId].radius;
									Real inter_dist = spheres[tId].radius + (proj_pos - spheres[tId].center).norm();

									//nbr_out.setElement(tId, cnt, j);
									//int idx_con = nbr_out.getElementIndex(tId, cnt);
									//list_j.insert(j);
									int idx_con = prefix[tId] + cnt;
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
									cnt++;
								}
								else if ((proj_pos - spheres[tId].center).norm() < spheres[tId].radius + boundary_expand
									&& (((proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm()).cross(t3d_tmp.normal() / t3d_tmp.normal().norm())).norm() < EPSILON)
								{
									//printf("YYEESS\n");
										//printf("========vvvvvv======\n");
									Coord3D inter_norm = (proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
									Coord3D p1 = proj_pos;
									Coord3D p2 = spheres[tId].center + inter_norm * spheres[tId].radius;
									Real inter_dist = spheres[tId].radius - (proj_pos - spheres[tId].center).norm();

									if (inter_dist < 0) inter_dist = 0;

									//nbr_out.setElement(tId, cnt, j);
									//int idx_con = nbr_out.getElementIndex(tId, cnt);
									//list_j.insert(j);
									int idx_con = prefix[tId] + cnt;
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_STICKINESS, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
									cnt++;
								}
							}
							else
							{
								int idx;
								Real max_dist = 0.0f;
								for (int iii = 0; iii < 4; iii++)
								{
									if (tets_sdf[4 * (j - elementOffset.tetOffset) + iii] > max_dist)
									{
										max_dist = tets_sdf[4 * (j - elementOffset.tetOffset) + iii];
										idx = iii;
									}
								}
								int id1 = (idx + 1) % 4;
								int id2 = (idx + 2) % 4;
								int id3 = (idx + 3) % 4;
								Triangle3D t3d(
									tets[j - elementOffset.tetOffset].v[id1],
									tets[j - elementOffset.tetOffset].v[id2],
									tets[j - elementOffset.tetOffset].v[id3]);
								proj_pos = pos_sphere.project(t3d).origin;

								if (pos_sphere.inside(tets[j - elementOffset.tetOffset]))
								{


									Coord3D inter_norm = -(proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
									Coord3D p1 = proj_pos;
									Coord3D p2 = spheres[tId].center - inter_norm * spheres[tId].radius;
									Real inter_dist = spheres[tId].radius + (proj_pos - spheres[tId].center).norm();

									//nbr_out.setElement(tId, cnt, j);
									//int idx_con = nbr_out.getElementIndex(tId, cnt);
									//list_j.insert(j);
									int idx_con = prefix[tId] + cnt;
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
									cnt++;
								}
							}
						}
						else
						{
							if (pos_sphere.inside(tets[j - elementOffset.tetOffset]))
							{

								Coord3D inter_norm = -(proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
								Coord3D p1 = proj_pos;
								Coord3D p2 = spheres[tId].center - inter_norm * spheres[tId].radius;
								Real inter_dist = spheres[tId].radius + (proj_pos - spheres[tId].center).norm();

								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = inter_dist;
								cnt++;
							}
							else if ((proj_pos - spheres[tId].center).norm() < spheres[tId].radius + boundary_expand)
							{
								Coord3D inter_norm = (proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
								Coord3D p1 = proj_pos;
								Coord3D p2 = spheres[tId].center + inter_norm * spheres[tId].radius;
								Real inter_dist = spheres[tId].radius - (proj_pos - spheres[tId].center).norm();
								if (inter_dist < 0) inter_dist = 0;

								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = inter_dist;
								cnt++;
							}
						}
					}

					break;
				}
				case CT_SEG:
				{
					if (filter.sphere_capsule)
					{
						Point3D pos_sphere(spheres[tId].center);
						Segment3D segment_tmp = caps[j - elementOffset.segOffset].segment;
						//printf("ssssssssssssssssssssaaaaaaaaaaaaaaaaa\n");
						if (pos_sphere.distance(segment_tmp) < spheres[tId].radius + caps[j - elementOffset.segOffset].radius + boundary_expand)
						{

							//printf("ssssssssssssssssssss\n");
							//printf("CNT_OKKKK\n");
							/*if( ((j - elementOffset.segOffset) % 39 == 0 || (j - elementOffset.segOffset) % 39 == 38)
								||
							((pos_sphere.project(segment_tmp).origin - pos_sphere.origin).dot(segment_tmp.direction()) < EPSILON))
							*/ {
								Coord3D proj_pos = pos_sphere.project(segment_tmp).origin;
								Coord3D inter_norm = (proj_pos - spheres[tId].center) / (proj_pos - spheres[tId].center).norm();
								Coord3D p1 = proj_pos - inter_norm * caps[j - elementOffset.segOffset].radius;
								Coord3D p2 = spheres[tId].center + inter_norm * spheres[tId].radius;
								Real inter_dist =
									spheres[tId].radius
									+ caps[j - elementOffset.segOffset].radius
									//+ boundary_expand
									- (proj_pos - spheres[tId].center).norm();

								if (inter_dist < 0) inter_dist = 0;

								/*printf("%.5lf %.5lf %.5lf     %.5lf\n",
									inter_norm[0], inter_norm[1], inter_norm[2],
									inter_dist);*/

								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								if (inter_dist > spheres[tId].radius)
								{
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist - spheres[tId].radius;
								}
								else
								{
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_STICKINESS, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
								}
								cnt++;
							}
						}
					}
					break;
				}
				case CT_TRI:
				{

					Point3D pos_sphere(spheres[tId].center);
					Triangle3D tri_tmp = tris[j - elementOffset.triOffset];
					Real proj_dist = abs(pos_sphere.distance(tri_tmp));
					if (filter.sphere_tri)
						if (proj_dist < spheres[tId].radius + 1.75 * boundary_expand && proj_dist > EPSILON
							//&& ((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).dot(tri_tmp.normal() / tri_tmp.normal().norm()) < EPSILON
							&& (((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).cross(tri_tmp.normal() / tri_tmp.normal().norm())).norm() < 0.001
							)
						{

							//printf();

							Coord3D proj_pos = pos_sphere.project(tri_tmp).origin;

							Coord3D inter_norm = (proj_pos - spheres[tId].center) / proj_dist;
							Coord3D p1 = proj_pos;
							Coord3D p2 = spheres[tId].center;// + inter_norm * spheres[tId].radius;
							Real inter_dist = spheres[tId].radius - proj_dist + boundary_expand * 1.25;
							if (inter_dist < 0) inter_dist = 0;

							//printf("%.3lf %.3lf %.3lf\n", inter_norm[0], inter_norm[1], inter_norm[2]);

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = inter_dist;
							if (inter_dist < boundary_expand * 0.25)
								nbr_cons[idx_con].contactType = ContactType::CT_FLUID_STICKINESS;
							cnt++;
							//cnt++;
						}
					break;

				}
				default:
					break;
				}
			}
			break;
		}
		case CT_BOX:
		{
			//printf("YES\n");
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			//List<int>& list_j = nbr_out[tId];
			int nbSize = list_i.size();
			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];
				ElementType eleType_j = checkElementType(j, elementOffset);
				/*if (j < 8)
				{
					printf("===================== !!!!!!!!!!!!! %d %d\n", tId, j);
				}*/
				switch (eleType_j)
				{
				case CT_SPHERE:
				{
					if (filter.sphere_box)
					{
						Point3D pos_sphere(spheres[j].center);
						Coord3D proj_pos = pos_sphere.project(boxes[tId - elementOffset.boxOffset]).origin;
						//printf("sphere!!!! === %d\n", j);
						if (pos_sphere.inside(boxes[tId - elementOffset.boxOffset]))
						{
							Coord3D inter_norm = -(proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
							Coord3D p1 = proj_pos;
							Coord3D p2 = spheres[j].center - inter_norm * spheres[j].radius;
							Real inter_dist = spheres[j].radius + (proj_pos - spheres[j].center).norm();

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							//nbr_cons[idx_con] = NeighborConstraints(tId, j, 1, p1, p2, 0.0f, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = inter_dist;
							cnt++;
							//printf("sphere!!!! %d\n", j);
						}
						else if ((proj_pos - spheres[j].center).norm() < spheres[j].radius + boundary_expand)
						{
							Coord3D inter_norm = (proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
							Coord3D p1 = proj_pos;
							Coord3D p2 = spheres[j].center + inter_norm * spheres[j].radius;
							Real inter_dist = spheres[j].radius - (proj_pos - spheres[j].center).norm();
							if (inter_dist < 0) inter_dist = 0;

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_STICKINESS, p1, p2, inter_norm, Coord3D(0, 0, 0));
							//nbr_cons[idx_con] = NeighborConstraints(tId, j, 1, p1, p2, 0.0f, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = 0.0f;//inter_dist;
							cnt++;
							//printf("sphere!!!! %d\n", j);
						}
					}
					break;
				}

				case CT_BOX: //boxes
				{
					TManifold<Real> manifold;

					auto boxA = boxes[tId - elementOffset.boxOffset];
					auto boxB = boxes[j - elementOffset.boxOffset];
					CollisionDetection<Real>::request(manifold, boxA, boxB);

					for (int cn = 0; cn < manifold.contactCount; cn++)
					{
						int idx_con = prefix[tId] + cnt;

						NeighborConstraints cPair;

						cPair.pos1 = manifold.contacts[cn].position;
						cPair.pos2 = manifold.contacts[cn].position;
						cPair.normal1 = -manifold.normal;
						cPair.normal2 = manifold.normal;
						cPair.bodyId1 = tId;
						cPair.bodyId2 = j;
						cPair.contactType = ContactType::CT_NONPENETRATION;
						cPair.interpenetration = -manifold.contacts[cn].penetration;
						nbr_cons[idx_con] = cPair;

						cnt += 1;
					}

// 					if (filter.box_box)
// 					{
// 						Coord3D inter_norm1, p11, p21;
// 						Coord3D inter_norm2, p12, p22;
// 						Real inter_dist1;
// 						Real inter_dist2;
// 
// 
// 						int type = 0;
// 						bool insert_one = boxes[tId - elementOffset.boxOffset].point_intersect(boxes[j - elementOffset.boxOffset], inter_norm1, inter_dist1, p11, p21);
// 						bool insert_two = boxes[j - elementOffset.boxOffset].point_intersect(boxes[tId - elementOffset.boxOffset], inter_norm2, inter_dist2, p12, p22);
// 
// 						if (insert_one && insert_two)
// 						{
// 							if (inter_dist1 < inter_dist2) type = 1;
// 							else type = 2;
// 						}
// 						else if (insert_one) type = 1;
// 						else if (insert_two) type = 2;
// 
// 						if (type == 1)
// 						{
// 							//nbr_out.setElement(tId, cnt, j);
// 
// 							/*set up constraints*/
// 							//int idx_con = nbr_out.getElementIndex(tId, cnt);
// 							//list_j.insert(j);
// 							int idx_con = prefix[tId] + cnt;
// 							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p11, p21, inter_norm1, Coord3D(0, 0, 0));
// 							nbr_cons[idx_con].interpenetration = -inter_dist1;
// 
// 							//printf("Interpenetration: %f \n", nbr_cons[idx_con].interpenetration);
// 							cnt++;
// 						}
// 						else if (type == 2)
// 						{
// 
// 							//nbr_out.setElement(tId, cnt, j);
// 
// 							/*set up constraints*/
// 							//int idx_con = nbr_out.getElementIndex(tId, cnt);
// 							//list_j.insert(j);
// 							int idx_con = prefix[tId] + cnt;
// 							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p12, p22, inter_norm2, Coord3D(0, 0, 0));
// 							nbr_cons[idx_con].interpenetration = -inter_dist2;
// 
// 							//printf("Interpenetration: %f \n", nbr_cons[idx_con].interpenetration);
// 
// 							cnt++;
// 						}
// 					}
					break;
				}
				case CT_TET:// tets
				{
					//printf("BBBBBBBBBBBBBBBBBBTTTTTTTTTTT\n");
					if (filter.box_tet)
					{
						Coord3D inter_norm, p1, p2;
						Real inter_dist;

						if (boxes[tId - elementOffset.boxOffset].point_intersect(tets[j - elementOffset.tetOffset], inter_norm, inter_dist, p1, p2))
						{
							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = -inter_dist;
							cnt++;
						}
					}
					break;
				}
				case CT_SEG:// segments
				{
					//printf("YYYYYYYYYEEEEEEEEESSSSSSSSSSSSS %d", j);
					if (filter.box_capsule)
					{
						Segment3D segment_tmp = caps[j - elementOffset.segOffset].segment;
						Segment3D segment_intersect;
						Segment3D segment_prox = segment_tmp.proximity(boxes[tId - elementOffset.boxOffset]);
						if (segment_tmp.intersect(boxes[tId - elementOffset.boxOffset], segment_intersect))
						{
							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							Coord3D p1, p2;
							Real interDist = 0.0f;
							Point3D inp((segment_intersect.startPoint() + segment_intersect.endPoint()) / 2.0f);
							Point3D sp(segment_intersect.startPoint());
							Point3D ep(segment_intersect.endPoint());
							if (abs(inp.distance(boxes[tId - elementOffset.boxOffset])) > abs(interDist))
							{
								p2 = inp.origin;
								p1 = inp.project(boxes[tId - elementOffset.boxOffset]).origin;
								interDist = -abs(inp.distance(boxes[tId - elementOffset.boxOffset]));
							}
							if (abs(sp.distance(boxes[tId - elementOffset.boxOffset])) > abs(interDist))
							{
								p2 = sp.origin;
								p1 = sp.project(boxes[tId - elementOffset.boxOffset]).origin;
								interDist = -abs(sp.distance(boxes[tId - elementOffset.boxOffset]));
							}
							if (abs(ep.distance(boxes[tId - elementOffset.boxOffset])) > abs(interDist))
							{
								p2 = ep.origin;
								p1 = ep.project(boxes[tId - elementOffset.boxOffset]).origin;
								interDist = -abs(ep.distance(boxes[tId - elementOffset.boxOffset]));
							}

							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, (p2 - p1) / (p1 - p2).norm(), Coord3D(0, 0, 0));


							nbr_cons[idx_con].interpenetration = -interDist + caps[j - elementOffset.segOffset].radius;
							
							cnt++;
						}
						else if (segment_prox.length() < caps[j - elementOffset.segOffset].radius)
						{
							//if (segment_prox.direction().dot(segment_tmp.direction()) < EPSILON)//////to delete
							{
								//nbr_out.setElement(tId, cnt, j);
								Coord3D p1, p2;
								Real interDist = caps[j - elementOffset.segOffset].radius - segment_prox.length();
								p1 = segment_prox.v1;
								p2 = segment_prox.v0;

								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, (p1 - p2) / (p1 - p2).norm(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = interDist;
								cnt++;
							}
						}
					}
					break;
				}
				case CT_TRI:// tris
				{
					//printf("BBBBBBBBBBBBBBBBBBTTTTTTTTTTT\n");
					if (filter.box_tri)
					{
						Coord3D inter_norm, p1, p2;
						Real inter_dist;
						//printf("AABox!!!!!!!!!!!!!!!!!!!\n");
						if (boxes[tId - elementOffset.boxOffset].point_intersect(tris[j - elementOffset.triOffset],
							inter_norm, inter_dist, p1, p2))
						{
							//printf("ABox!!!!!!!!!!!!!!!!!!!\n");
							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = -inter_dist;
							cnt++;
						}
					}
					break;
				}
				default:
				{
					break;
				}
				}
			}
			break;
		}

		case CT_TET:
		{
			//printf("YYYYYYYYYYYEEEEEEEEEEESSSSSSSSSSS\n");
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			//List<int>& list_j = nbr_out[tId];
			int nbSize = list_i.size();

			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];
				ElementType eleType_j = checkElementType(j, elementOffset);
				switch (eleType_j)
				{
				case CT_SPHERE:
				{
					if (filter.sphere_tet)
					{
						if (filter.tet_sdf)// && tets_sdf.size() > 0)
						{
							Point3D pos_sphere(spheres[j].center);
							Bool tmp;
							int idx;
							Coord3D proj_pos = pos_sphere.project(tets[tId - elementOffset.tetOffset], tmp, &idx).origin;

							int id1 = (idx + 1) % 4;
							int id2 = (idx + 2) % 4;
							int id3 = (idx + 3) % 4;

							Triangle3D t3d_tmp = Triangle3D(
								tets[tId - elementOffset.tetOffset].v[id1],
								tets[tId - elementOffset.tetOffset].v[id2],
								tets[tId - elementOffset.tetOffset].v[id3]
							);

							if (tets_sdf[4 * (tId - elementOffset.tetOffset) + id1] < EPSILON &&
								tets_sdf[4 * (tId - elementOffset.tetOffset) + id2] < EPSILON &&
								tets_sdf[4 * (tId - elementOffset.tetOffset) + id3] < EPSILON)
							{
								if (pos_sphere.inside(tets[tId - elementOffset.tetOffset]))
								{
									Coord3D inter_norm = -(proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
									Coord3D p1 = proj_pos;
									Coord3D p2 = spheres[j].center - inter_norm * spheres[j].radius;
									Real inter_dist = spheres[j].radius + (proj_pos - spheres[j].center).norm();

									//nbr_out.setElement(tId, cnt, j);
									//int idx_con = nbr_out.getElementIndex(tId, cnt);
									//list_j.insert(j);
									int idx_con = prefix[tId] + cnt;
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
									cnt++;
								}
								else if ((proj_pos - spheres[j].center).norm() < spheres[j].radius + boundary_expand
									&& (((proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm()).cross(t3d_tmp.normal() / t3d_tmp.normal().norm())).norm() < EPSILON)
								{
									//printf("YYEESS\n");
									Coord3D inter_norm = (proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
									Coord3D p1 = proj_pos;
									Coord3D p2 = spheres[j].center + inter_norm * spheres[j].radius;
									Real inter_dist = spheres[j].radius - (proj_pos - spheres[j].center).norm();
									if (inter_dist < 0) inter_dist = 0;

									//nbr_out.setElement(tId, cnt, j);
									//int idx_con = nbr_out.getElementIndex(tId, cnt);
									//list_j.insert(j);
									int idx_con = prefix[tId] + cnt;
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_STICKINESS, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
									cnt++;
								}
							}
							else
							{
								int idx;
								Real max_dist = 0.0f;
								for (int iii = 0; iii < 4; iii++)
								{
									if (tets_sdf[4 * (tId - elementOffset.tetOffset) + iii] > max_dist)
									{
										max_dist = tets_sdf[4 * (tId - elementOffset.tetOffset) + iii];
										idx = iii;
									}
								}
								int id1 = (idx + 1) % 4;
								int id2 = (idx + 2) % 4;
								int id3 = (idx + 3) % 4;
								Triangle3D t3d(
									tets[j - elementOffset.tetOffset].v[id1],
									tets[j - elementOffset.tetOffset].v[id2],
									tets[j - elementOffset.tetOffset].v[id3]);
								proj_pos = pos_sphere.project(t3d).origin;
								if (pos_sphere.inside(tets[tId - elementOffset.tetOffset]))
								{
									Coord3D inter_norm = -(proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
									Coord3D p1 = proj_pos;
									Coord3D p2 = spheres[j].center - inter_norm * spheres[j].radius;
									Real inter_dist = spheres[j].radius + (proj_pos - spheres[j].center).norm();

									//nbr_out.setElement(tId, cnt, j);
									//int idx_con = nbr_out.getElementIndex(tId, cnt);
									//list_j.insert(j);
									int idx_con = prefix[tId] + cnt;
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;
									cnt++;
								}
							}
						}
						else
						{
							Point3D pos_sphere(spheres[j].center);
							Bool tmp;
							int idx;
							Coord3D proj_pos = pos_sphere.project(tets[tId - elementOffset.tetOffset], tmp, &idx).origin;
							if (pos_sphere.inside(tets[tId - elementOffset.tetOffset]))
							{
								Coord3D inter_norm = -(proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
								Coord3D p1 = proj_pos;
								Coord3D p2 = spheres[j].center - inter_norm * spheres[j].radius;
								Real inter_dist = spheres[j].radius + (proj_pos - spheres[j].center).norm();

								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = inter_dist;
								cnt++;
							}
							else if ((proj_pos - spheres[j].center).norm() < spheres[j].radius + boundary_expand)
							{
								Coord3D inter_norm = (proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
								Coord3D p1 = proj_pos;
								Coord3D p2 = spheres[j].center + inter_norm * spheres[j].radius;
								Real inter_dist = spheres[j].radius - (proj_pos - spheres[j].center).norm();
								if (inter_dist < 0) inter_dist = 0;

								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = inter_dist;
								cnt++;
							}
						}
					}
					break;
				}
				case CT_BOX:
				{

					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.box_tet)
						if (boxes[j - elementOffset.boxOffset].point_intersect(tets[tId - elementOffset.tetOffset], inter_norm, inter_dist, p1, p2))
						{

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = -inter_dist;
							cnt++;
							//printf("TTTTTTTTTTTBBBBBBBBBBBBBBBBBB interDist = %.3lf   interNormal = %.3lf %.3lf %.3lf\n",
							//inter_dist,
							//inter_norm[0], inter_norm[1], inter_norm[2]);
						}

					break;
				}
				case CT_TET:
				{
					if (filter.tet_tet)
					{

						if (filter.tet_neighbor_filter
							&& tet_body_ids.size() > 0
							&& tet_body_ids[tId - elementOffset.tetOffset] == tet_body_ids[j - elementOffset.tetOffset])
						{
							bool tmp = false;
							for (int iii = 0; iii < 4; iii++)
							{ 
								for (int jjj = 0; jjj < 4; jjj++)
									if (tet_element_ids[tId - elementOffset.tetOffset][iii]
										== tet_element_ids[j - elementOffset.tetOffset][jjj])
									{
										tmp = true;
										break;
									}
								if (tmp)
									break;
							}
							if (tmp)
								break;

						}

						Coord3D inter_norm1, p11, p21;
						Coord3D inter_norm2, p12, p22;
						Real inter_dist1;
						Real inter_dist2;



						int type = 0;
						bool insert_one = tets[tId - elementOffset.tetOffset].intersect(tets[j - elementOffset.tetOffset], inter_norm1, inter_dist1, p11, p21);
						bool insert_two = tets[j - elementOffset.tetOffset].intersect(tets[tId - elementOffset.tetOffset], inter_norm2, inter_dist2, p12, p22);


						Coord3D tet_center1 = (tets[tId - elementOffset.tetOffset].v[0]
							+ tets[tId - elementOffset.tetOffset].v[1]
							+ tets[tId - elementOffset.tetOffset].v[2]
							+ tets[tId - elementOffset.tetOffset].v[3])
							/ 4.0f;
						Coord3D tet_center2 = (tets[j - elementOffset.tetOffset].v[0]
							+ tets[j - elementOffset.tetOffset].v[1]
							+ tets[j - elementOffset.tetOffset].v[2]
							+ tets[j - elementOffset.tetOffset].v[3])
							/ 4.0f;
						if (insert_one && insert_two)
						{
							if (inter_dist1 < inter_dist2 && inter_dist1 < - 3 * EPSILON) type = 1;
							else if(inter_dist2 < - 3 * EPSILON) type = 2;
						}
						else if (insert_one && inter_dist1 < -3 * EPSILON) type = 1;
						else if (insert_two && inter_dist2 < -3 * EPSILON) type = 2;

						ContactType ctype = ContactType::CT_FLUID_NONPENETRATION;

						if (type == 1)
						{
							//list_j.insert(j);
							//int idx_con = prefix[tId] + cnt;
							/*set up constraints*/
							if (abs(inter_dist1) < EPSILON)
							{
								inter_norm1 = (tet_center1 - tet_center2) / (tet_center1 - tet_center2).norm();
								ctype = ContactType::CT_LOACL_NONPENETRATION;
							}

							int idx;
							Real max_dist = 0.0f;
							for (int iii = 0; iii < 4; iii++)
							{
								
								if (tets_sdf.size() > 0)
									if (tets_sdf[4 * (j - elementOffset.tetOffset) + iii] > max_dist)
									{

										max_dist = tets_sdf[4 * (j - elementOffset.tetOffset) + iii];
										idx = iii;
									}
							}
							int id1 = (idx + 1) % 4;
							int id2 = (idx + 2) % 4;
							int id3 = (idx + 3) % 4;
							Triangle3D t3d(
								tets[j - elementOffset.tetOffset].v[id1],
								tets[j - elementOffset.tetOffset].v[id2],
								tets[j - elementOffset.tetOffset].v[id3]);
							Coord3D proj_pos = Point3D(p11).project(t3d).origin;


							if (max_dist < EPSILON || abs(inter_dist1) < EPSILON)
							{ 
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								if (idx_con < nbr_cons.size())
								{
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ctype, p11, p21, inter_norm1, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = -inter_dist1;
								}
							}
							else
							{
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								if (idx_con < nbr_cons.size())
								{
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ctype, p11, proj_pos,
										(proj_pos - p11) / (proj_pos - p11).norm(), Coord3D(0, 0, 0));

									nbr_cons[idx_con].interpenetration = (proj_pos - p11).norm();
								}
							}

							cnt++;

						}
						else if (type == 2)
						{
							//list_j.insert(j);
							//int idx_con = prefix[tId] + cnt;
							if (abs(inter_dist2) < EPSILON)
							{
								inter_norm2 = (tet_center2 - tet_center1) / (tet_center1 - tet_center2).norm();
								ctype = ContactType::CT_LOACL_NONPENETRATION;
							}
							/*set up constraints*/
							int idx;
							Real max_dist = 0.0f;
							for (int iii = 0; iii < 4; iii++)
							{
								if(tets_sdf.size() > 0)
									if (tets_sdf[4 * (tId - elementOffset.tetOffset) + iii] > max_dist)
									{
										max_dist = tets_sdf[4 * (tId - elementOffset.tetOffset) + iii];
										idx = iii;
									}
							}
							int id1 = (idx + 1) % 4;
							int id2 = (idx + 2) % 4;
							int id3 = (idx + 3) % 4;
							Triangle3D t3d(
								tets[tId - elementOffset.tetOffset].v[id1],
								tets[tId - elementOffset.tetOffset].v[id2],
								tets[tId - elementOffset.tetOffset].v[id3]);
							Coord3D proj_pos = Point3D(p12).project(t3d).origin;


							if (max_dist < EPSILON || abs(inter_dist1) < EPSILON)
							{
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								if(idx_con < nbr_cons.size())
								{ 
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ctype, p12, p22, inter_norm2, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = -inter_dist2;
								}
								//printf("%.5lf\n", inter_dist2);
							}
							else
							{
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;
								if(idx_con < nbr_cons.size())
								{ 
									nbr_cons[idx_con] = NeighborConstraints(j, tId, ctype, p12, proj_pos,
										(proj_pos - p12) / (proj_pos - p12).norm(), Coord3D(0, 0, 0));

									nbr_cons[idx_con].interpenetration = (proj_pos - p12).norm();
								}
							}
							cnt++;
						}
						
					}
					break;
				}
				case CT_SEG:
				{
					if (filter.tet_capsule)
					{
						Segment3D segment_tmp = caps[j - elementOffset.segOffset].segment;
						Line3D line_tmp(segment_tmp.v0, segment_tmp.direction());
						Segment3D segment_tmp2 = segment_tmp.proximity(tets[tId - elementOffset.tetOffset]);
						Segment3D segment_intersect;
						bool intersect1 = false;
						if (line_tmp.intersect(tets[tId - elementOffset.tetOffset], segment_intersect))
						{
							Real left = (segment_intersect.v0 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							Real right = (segment_intersect.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < left)
							{
								Real tmp = left;
								left = right;
								right = tmp;
							}
							Real maxx = (segment_tmp.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < 0 || left > maxx)
							{

							}
							else
							{
								intersect1 = true;
								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								left = max(left, 0.0f);
								right = min(right, maxx);


								Coord3D p1, p2;
								Real interDist = 0.0f;
								Bool tmp_bool;

								Coord3D p11 = segment_tmp.v0 + ((left + right) / 2.0f * segment_tmp.direction().normalize());
								Coord3D p22 = Point3D(p11).project(tets[tId - elementOffset.tetOffset], tmp_bool).origin;//


								if ((p11 - p22).norm() > abs(interDist))
								{
									interDist = -(p11 - p22).norm();
									p1 = p11;
									p2 = p22;
								}
								p11 = segment_tmp.v0 + left * segment_tmp.direction().normalize();
								p22 = Point3D(p11).project(tets[tId - elementOffset.tetOffset], tmp_bool).origin;
								if ((p11 - p22).norm() > abs(interDist))
								{
									p1 = p11; p2 = p22;
									interDist = -(p1 - p2).norm();
								}

								p11 = segment_tmp.v0 + right * segment_tmp.direction().normalize();
								p22 = Point3D(p11).project(tets[tId - elementOffset.tetOffset], tmp_bool).origin;
								if ((p11 - p22).norm() > abs(interDist))
								{
									p1 = p11; p2 = p22;
									interDist = -(p1 - p2).norm();
								}

								nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, (p2 - p1) / (p1 - p2).norm(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = -interDist + caps[j - elementOffset.segOffset].radius;
								cnt++;
							}
						}
						if (!intersect1)
						{
							if (segment_tmp2.length() < caps[j - elementOffset.segOffset].radius)
							{
								//nbr_out.setElement(tId, cnt, j);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								Coord3D p1, p2;
								Real interDist = caps[j - elementOffset.segOffset].radius - segment_tmp2.length();
								p1 = segment_tmp2.v1;
								p2 = segment_tmp2.v0;

								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, (p1 - p2) / (p1 - p2).norm(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = interDist;
								cnt++;
							}
						}
					}
					break;
				}
				case CT_TRI:
				{

					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.tet_tri)
						if (tets[tId - elementOffset.boxOffset].intersect(tris[j - elementOffset.triOffset], inter_norm,
							inter_dist, p1, p2))
						{

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = -inter_dist;
							cnt++;

						}

					break;
				}
				default:
				{
					break;
				}
				}

			}
			break;
		}
		case CT_SEG:
		{
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			//List<int>& list_j = nbr_out[tId];
			int nbSize = list_i.size();
			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);

				switch (eleType_j)
				{
				case CT_SPHERE:
				{
					if (filter.sphere_capsule)
					{
						Point3D pos_sphere(spheres[j].center);
						Segment3D segment_tmp = caps[tId - elementOffset.segOffset].segment;

						if (pos_sphere.distance(segment_tmp) < spheres[j].radius + caps[tId - elementOffset.segOffset].radius + boundary_expand)
						{

							//printf("%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%\n");
							/*if (((tId - elementOffset.segOffset) % 39 == 0 || (tId - elementOffset.segOffset) % 39 == 38)
								||
								((pos_sphere.project(segment_tmp).origin - pos_sphere.origin).dot(segment_tmp.direction()) < EPSILON))
							*/ {
								Coord3D proj_pos = pos_sphere.project(segment_tmp).origin;
								Coord3D inter_norm = (proj_pos - spheres[j].center) / (proj_pos - spheres[j].center).norm();
								Coord3D p1 = proj_pos - inter_norm * caps[tId - elementOffset.segOffset].radius;
								Coord3D p2 = spheres[j].center + inter_norm * spheres[j].radius;
								Real inter_dist =
									spheres[j].radius
									+ caps[tId - elementOffset.segOffset].radius
									- (proj_pos - spheres[j].center).norm();

								if (inter_dist < 0) inter_dist = 0;
								/*printf("%.5lf %.5lf %.5lf     %.5lf\n",
									inter_norm[0], inter_norm[1], inter_norm[2],
									inter_dist);*/

								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								if (inter_dist > spheres[j].radius)
								{
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist - spheres[j].radius;//inter_dist;
								}
								else
								{
									nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_STICKINESS, p1, p2, inter_norm, Coord3D(0, 0, 0));
									nbr_cons[idx_con].interpenetration = inter_dist;//inter_dist;
								}
								cnt++;
							}
						}
					}
					break;
				}
				case CT_BOX:// segments
				{
					//printf("YYYYYYYYYEEEEEEEEESSSSSSSSSSSSS2222 %d", j);
					if (filter.box_capsule)
					{
						Segment3D segment_tmp = caps[tId - elementOffset.segOffset].segment;
						Segment3D segment_intersect;
						Segment3D segment_prox = segment_tmp.proximity(boxes[j - elementOffset.boxOffset]);
						if (segment_tmp.intersect(boxes[j - elementOffset.boxOffset], segment_intersect))
						{
							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							Coord3D p1, p2;
							Real interDist = 0.0f;
							Point3D inp((segment_intersect.startPoint() + segment_intersect.endPoint()) / 2.0f);
							Point3D sp(segment_intersect.startPoint());
							Point3D ep(segment_intersect.endPoint());
							if (abs(inp.distance(boxes[j - elementOffset.boxOffset])) > abs(interDist))
							{
								p2 = inp.origin;
								p1 = inp.project(boxes[j - elementOffset.boxOffset]).origin;
								interDist = -abs(inp.distance(boxes[j - elementOffset.boxOffset]));
							}
							if (abs(sp.distance(boxes[j - elementOffset.boxOffset])) > abs(interDist))
							{
								p2 = sp.origin;
								p1 = sp.project(boxes[j - elementOffset.boxOffset]).origin;
								interDist = -abs(sp.distance(boxes[j - elementOffset.boxOffset]));
							}
							if (abs(ep.distance(boxes[j - elementOffset.boxOffset])) > abs(interDist))
							{
								p2 = ep.origin;
								p1 = ep.project(boxes[j - elementOffset.boxOffset]).origin;
								interDist = -abs(ep.distance(boxes[j - elementOffset.boxOffset]));
							}

							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, (p2 - p1) / (p1 - p2).norm(), Coord3D(0, 0, 0));


							nbr_cons[idx_con].interpenetration = -interDist + caps[tId - elementOffset.segOffset].radius;
							/*printf(" ================ %d %d %.13lf %.13lf %.13lf   %.13lf %.13lf %.13lf %.13lf\n",
								tId, j,
								segment_tmp.v0[0], segment_tmp.v0[1], segment_tmp.v0[2],
								segment_tmp.v1[0], segment_tmp.v1[1], segment_tmp.v1[2],
								segment_intersect.length());*/
							cnt++;
						}
						else if (segment_prox.length() < caps[tId - elementOffset.segOffset].radius)
						{
							//if (segment_prox.direction().dot(segment_tmp.direction()) < EPSILON)//////to delete
							{
								//nbr_out.setElement(tId, cnt, j);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								Coord3D p1, p2;
								Real interDist = caps[tId - elementOffset.segOffset].radius - segment_prox.length();
								p1 = segment_prox.v1;
								p2 = segment_prox.v0;

								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, (p1 - p2) / (p1 - p2).norm(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = interDist;
								cnt++;
							}
						}
					}
					break;
				}
				case CT_TET:
				{
					if (filter.tet_capsule)
					{
						Segment3D segment_tmp = caps[tId - elementOffset.segOffset].segment;
						Line3D line_tmp(segment_tmp.v0, segment_tmp.direction());
						Segment3D segment_tmp2 = segment_tmp.proximity(tets[j - elementOffset.tetOffset]);
						Segment3D segment_intersect;
						bool intersect1 = false;
						if (line_tmp.intersect(tets[j - elementOffset.tetOffset], segment_intersect))
						{
							Real left = (segment_intersect.v0 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							Real right = (segment_intersect.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < left)
							{
								Real tmp = left;
								left = right;
								right = tmp;
							}
							Real maxx = (segment_tmp.v1 - segment_tmp.v0).dot(segment_tmp.direction().normalize());
							if (right < 0 || left > maxx)
							{

							}
							else
							{
								intersect1 = true;
								//nbr_out.setElement(tId, cnt, j);
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								left = max(left, 0.0f);
								right = min(right, maxx);


								Coord3D p1, p2;
								Real interDist = 0.0f;
								Bool tmp_bool;

								Coord3D p11 = segment_tmp.v0 + ((left + right) / 2.0f * segment_tmp.direction().normalize());
								Coord3D p22 = Point3D(p11).project(tets[j - elementOffset.tetOffset], tmp_bool).origin;//


								if ((p11 - p22).norm() > abs(interDist))
								{
									interDist = -(p11 - p22).norm();
									p1 = p11;
									p2 = p22;
								}
								p11 = segment_tmp.v0 + left * segment_tmp.direction().normalize();
								p22 = Point3D(p11).project(tets[j - elementOffset.tetOffset], tmp_bool).origin;
								if ((p11 - p22).norm() > abs(interDist))
								{
									p1 = p11; p2 = p22;
									interDist = -(p1 - p2).norm();
								}

								p11 = segment_tmp.v0 + right * segment_tmp.direction().normalize();
								p22 = Point3D(p11).project(tets[j - elementOffset.tetOffset], tmp_bool).origin;
								if ((p11 - p22).norm() > abs(interDist))
								{
									p1 = p11; p2 = p22;
									interDist = -(p1 - p2).norm();
								}

								nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_NONPENETRATION, p1, p2, (p2 - p1) / (p1 - p2).norm(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = -interDist + caps[tId - elementOffset.segOffset].radius;
								cnt++;
							}
						}
						if (!intersect1)
						{
							if (segment_tmp2.length() < caps[tId - elementOffset.segOffset].radius)
							{
								//nbr_out.setElement(tId, cnt, j);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								Coord3D p1, p2;
								Real interDist = caps[tId - elementOffset.segOffset].radius - segment_tmp2.length();
								p1 = segment_tmp2.v1;
								p2 = segment_tmp2.v0;

								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, (p1 - p2) / (p1 - p2).norm(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = interDist;
								cnt++;
							}
						}
					}
					break;
				}
				case CT_SEG:
				{
					if (filter.capsule_capsule)
					{
						Segment3D segment_1 = caps[j - elementOffset.segOffset].segment;
						Segment3D segment_2 = caps[tId - elementOffset.segOffset].segment;
						Segment3D proxi = (segment_1.proximity(segment_2));
						if (abs(j - tId) >= 3)
							if (proxi.length() < caps[j - elementOffset.segOffset].radius + caps[tId - elementOffset.segOffset].radius)
							{
								Coord3D p1, p2;
								Real interDist = caps[j - elementOffset.segOffset].radius + caps[tId - elementOffset.segOffset].radius - proxi.length();
								p1 = proxi.v1 - proxi.direction() / proxi.length() * caps[tId - elementOffset.segOffset].radius;
								p2 = proxi.v0 + proxi.direction() / proxi.length() * caps[j - elementOffset.segOffset].radius;
								//????????????????????
								//int idx_con = nbr_out.getElementIndex(tId, cnt);
								//list_j.insert(j);
								int idx_con = prefix[tId] + cnt;

								nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, proxi.direction() / proxi.length(), Coord3D(0, 0, 0));
								nbr_cons[idx_con].interpenetration = interDist;
								//cnt++;
								cnt++;
							}
					}
					break;
				}
				case CT_TRI:
				{
					if (filter.capsule_tri)
					{
						Segment3D segment = caps[tId - elementOffset.segOffset].segment;
						Triangle3D triangle = tris[j - elementOffset.triOffset];
						Point3D p3d;
						if (segment.intersect(triangle, p3d))
						{
							Coord3D p1, p2;

							if (p3d.distance(Point3D(segment.v0)) < p3d.distance(Point3D(segment.v1)))
								p1 = segment.v0;
							else
								p1 = segment.v1;

							//p2 = p3d.origin;
							p2 = (Point3D(p1).project(triangle)).origin;

							Real interDist = (p1 - p2).norm();


							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;
							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, (p2 - p1) / interDist, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = 0;//interDist;
							cnt++;
						}
						else if ((segment.proximity(triangle)).length() < caps[tId - elementOffset.segOffset].radius)
						{
							//printf("bbbbbbbb\n");
							Segment3D proxi = segment.proximity(triangle);
							Coord3D p1, p2;
							Real interDist = caps[j - elementOffset.segOffset].radius - (segment.proximity(triangle)).length();
							p1 = proxi.v0;
							p2 = proxi.v1;
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, -proxi.direction() / proxi.length(), Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = 0;// interDist;
							cnt++;
						}
					}
					break;
				}
				default:
					break;
				}
			}
			break;
		}
		case CT_TRI:
		{

			//printf("TTTRRRRRRRRIIIIIIIII\n");
			//int nbSize = nbr.getNeighborSize(tId);
			List<int>& list_i = nbr[tId];
			//List<int>& list_j = nbr_out[tId];
			int nbSize = list_i.size();

			for (int ne = 0; ne < nbSize; ne++)
			{
				//int j = nbr.getElement(tId, ne);
				int j = list_i[ne];

				ElementType eleType_j = checkElementType(j, elementOffset);

				switch (eleType_j)
				{

				case CT_SPHERE:
				{
					Point3D pos_sphere(spheres[j].center);
					Triangle3D tri_tmp = tris[tId - elementOffset.triOffset];
					Real proj_dist = abs(pos_sphere.distance(tri_tmp));
					if (filter.sphere_tri)
						//if (pos_sphere.distance(tri_tmp) < spheres[j].radius && pos_sphere.distance(tri_tmp) > EPSILON)
						if (proj_dist < spheres[j].radius + 1.75 * boundary_expand && proj_dist > EPSILON
							//&& ((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).dot(tri_tmp.normal() / tri_tmp.normal().norm()) < EPSILON
							&& (((pos_sphere.project(tri_tmp).origin - pos_sphere.origin) / (pos_sphere.project(tri_tmp).origin - pos_sphere.origin).norm()).cross(tri_tmp.normal() / tri_tmp.normal().norm())).norm() < 0.001
							)
						{
							Coord3D proj_pos = pos_sphere.project(tri_tmp).origin;

							Coord3D inter_norm = (proj_pos - spheres[j].center) / proj_dist;
							Coord3D p1 = proj_pos;
							Coord3D p2 = spheres[j].center;// + inter_norm * spheres[j].radius;
							Real inter_dist = spheres[j].radius - proj_dist + boundary_expand * 1.25;
							if (inter_dist < 0) inter_dist = 0;

							//printf("%.3lf %.3lf %.3lf\n", inter_norm[0], inter_norm[1], inter_norm[2]);

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(tId, j, ContactType::CT_FLUID_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = inter_dist;
							if (inter_dist < boundary_expand * 0.25)
								nbr_cons[idx_con].contactType = ContactType::CT_FLUID_STICKINESS;
							cnt++;

						}
					break;
				}
				case CT_BOX:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.box_tri)
					{
						Coord3D inter_norm, p1, p2;
						Real inter_dist;
						//printf("BBBox!!!!!!!!!!!!!!!!!!!\n");
						if (boxes[j - elementOffset.boxOffset].point_intersect(tris[tId - elementOffset.triOffset],
							inter_norm, inter_dist, p1, p2))
						{
							//printf("BBox!!!!!!!!!!!!!!!!!!!\n");
							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);

							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = -inter_dist;
							cnt++;
						}
					}
					break;
				}
				case CT_TET:
				{
					Coord3D inter_norm, p1, p2;
					Real inter_dist;
					if (filter.tet_tri)
						if (tets[j - elementOffset.boxOffset].intersect(tris[tId - elementOffset.triOffset], inter_norm,
							inter_dist, p1, p2))
						{

							//nbr_out.setElement(tId, cnt, j);
							//int idx_con = nbr_out.getElementIndex(tId, cnt);

							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_NONPENETRATION, p1, p2, inter_norm, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = -inter_dist;
							cnt++;

						}
					break;
				}
				case CT_SEG:
				{

					////Segment3D segment_1 = caps[j - elementOffset.segOffset].segment;
					//Segment3D segment = caps[j - elementOffset.segOffset].segment;
					//Triangle3D triangle = triangles[tId - elementOffset.triOffset];
					//Point3D p3d;
					//if (segment.intersect(triangle, p3d))
					//{
					//	cnt++;
					//}
					//else if ((segment.proximity(triangle)).length() < caps[j - elementOffset.segOffset].radius)
					//{
					//	cnt++;
					//}
					if (filter.capsule_tri)
					{
						Segment3D segment = caps[j - elementOffset.segOffset].segment;
						Triangle3D triangle = tris[tId - elementOffset.triOffset];
						Point3D p3d;
						if (segment.intersect(triangle, p3d))
						{
							Coord3D p1, p2;

							if (p3d.distance(Point3D(segment.v0)) < p3d.distance(Point3D(segment.v1)))
								p1 = segment.v0;
							else
								p1 = segment.v1;

							p2 = (Point3D(p1).project(triangle)).origin;//p1.origin;

							Real interDist = (p1 - p2).norm();


							//int idx_con = nbr_out.getElementIndex(tId, cnt);
							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, (p2 - p1) / interDist, Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = 0; interDist;
							cnt++;
						}
						else if ((segment.proximity(triangle)).length() < caps[j - elementOffset.segOffset].radius)
						{
							//printf("aaaaaa\n");
							Segment3D proxi = segment.proximity(triangle);
							Coord3D p1, p2;
							Real interDist = caps[j - elementOffset.segOffset].radius - (segment.proximity(triangle)).length();
							p1 = proxi.v0;
							p2 = proxi.v1;
							//int idx_con = nbr_out.getElementIndex(tId, cnt);

							//list_j.insert(j);
							int idx_con = prefix[tId] + cnt;

							nbr_cons[idx_con] = NeighborConstraints(j, tId, ContactType::CT_FLUID_NONPENETRATION, p1, p2, -proxi.direction() / proxi.length(), Coord3D(0, 0, 0));
							nbr_cons[idx_con].interpenetration = 0;//interDist;
							cnt++;
						}
					}
					break;

				}
				case CT_TRI:
				{

					break;
				}
				default:
					break;
				}
			}
			break;
		}
		default:
			break;
		}
	}

	template<typename TDataType>
	void NeighborElementQuery<TDataType>::compute()
	{
		auto inTopo = this->inDiscreteElements()->getDataPtr();

		if (inTopo->totalSize() > 0)
		{
			Real boundary_expand = 0.0f;
			//printf("=========== ============= INSIDE SELF COLLISION %d\n", discreteSet->getTets().size());
			int t_num = inTopo->totalSize();
			if (m_queriedAABB.size() != t_num)
			{
				m_queriedAABB.resize(t_num);
			}
			if (m_queryAABB.size() != t_num)
			{
				m_queryAABB.resize(t_num);
			}

			ElementOffset elementOffset = inTopo->calculateElementOffset();

			cuExecute(t_num,
				NEQ_SetupAABB,
				m_queriedAABB,
				inTopo->getBoxes(),
				inTopo->getSpheres(),
				inTopo->getTets(),
				inTopo->getCaps(),
				inTopo->getTris(),
				elementOffset,
				boundary_expand);

			m_queryAABB.assign(m_queriedAABB);
			

			Real radius = this->inRadius()->getData();

			m_broadPhaseCD->varGridSizeLimit()->setValue(2 * radius);
			m_broadPhaseCD->setSelfCollision(true);


			/*if (this->outNeighborhood()->getElementCount() != t_num)
			{
				this->outNeighborhood()->setElementCount(t_num);
			}*/

			m_broadPhaseCD->inSource()->setValue(m_queryAABB);
			m_broadPhaseCD->inTarget()->setValue(m_queriedAABB);
			// 
			m_broadPhaseCD->update();
	
			Real zero = 0;
	
			//return;
			DArray<int> mapping_nbr;
			DArray<int> cnt_element;
			
			cnt_element.resize(inTopo->totalSize());
			cnt_element.reset();

			cuExecute(inTopo->totalSize(),
				NEQ_Narrow_Count,
				m_broadPhaseCD->outContactList()->getData(),
				inTopo->getBoxes(),
				inTopo->getSpheres(),
				inTopo->getTets(),
				inTopo->getTetSDF(),
				inTopo->getTetBodyMapping(),
				inTopo->getTetElementMapping(),
				inTopo->getCaps(),
				inTopo->getTris(),
				//nbrNum,
				cnt_element,
				elementOffset,
				Filter,
				boundary_expand);

			if (this->outContacts()->isEmpty())
				this->outContacts()->allocate();

			int sum = m_reduce.accumulate(cnt_element.begin(), cnt_element.size());

			auto& contacts = this->outContacts()->getData();
			m_scan.exclusive(cnt_element, true);
			contacts.resize(sum);
			if (sum > 0)
			{
				cuExecute(inTopo->totalSize(),
					NEQ_Narrow_Set,
					m_broadPhaseCD->outContactList()->getData(),
					inTopo->getBoxes(),
					inTopo->getSpheres(),
					inTopo->getTets(),
					inTopo->getTetSDF(),
					inTopo->getTetBodyMapping(),
					inTopo->getTetElementMapping(),
					inTopo->getCaps(),
					inTopo->getTris(),
					contacts,
					cnt_element,
					elementOffset,
					Filter,
					boundary_expand
				);
			}

			mapping_nbr.clear();
			cnt_element.clear();

		}
		else
		{
			//printf("NeighborElementQuery: Empty discreteSet! \n");
		}
	}

	DEFINE_CLASS(NeighborElementQuery);
}