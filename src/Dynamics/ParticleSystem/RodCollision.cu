#include "hip/hip_runtime.h"
#include "RodCollision.h"
#include "Framework/Node.h"
#include "Framework/CollidableObject.h"
#include "Collision/CollidablePoints.h"
#include "Topology/NeighborQuery.h"

namespace dyno
{
	IMPLEMENT_CLASS_1(RodCollision, TDataType)

	template<typename TDataType>
	RodCollision<TDataType>::RodCollision()
		: CollisionModel()
	{
	}

	template<typename TDataType>
	RodCollision<TDataType>::~RodCollision()
	{
		m_collidableObjects.clear();
	}

	template<typename TDataType>
	bool RodCollision<TDataType>::isSupport(std::shared_ptr<CollidableObject> obj)
	{
		if (obj->getType() == CollidableObject::POINTSET_TYPE)
		{
			return true;
		}
		return false;
	}


	template<typename TDataType>
	void RodCollision<TDataType>::addCollidableObject(std::shared_ptr<CollidableObject> obj)
	{
		auto derived = std::dynamic_pointer_cast<CollidablePoints<TDataType>>(obj);
		if (obj->getType() == CollidableObject::POINTSET_TYPE)
		{
			m_collidableObjects.push_back(derived);
		}
	}

	template<typename Real, typename Coord>
	__global__ void K_Collide(
		GArray<int> objIds,
		GArray<Coord> points,
		GArray<Coord> newPoints,
		GArray<Real> weights,
		NeighborList<int> neighbors,
		Real radius
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= points.size()) return;

		Real r;
		Coord pos_i = points[pId];
		int id_i = objIds[pId];
		int nbSize = neighbors.getNeighborSize(pId);
		int col_num = 0;
		Coord pos_num = Coord(0);
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors.getElement(pId, ne);
			r = (pos_i - points[j]).norm();
			if (r < radius && objIds[j] != id_i)
			{
				col_num++;
				Coord center = (pos_i + points[j]) / 2;
				Coord n = pos_i - center;
				if (n.norm() < EPSILON)
					n = Coord(1, 0, 0);
				else
				{
					n = n.normalize();
				}

				Coord target_i = (center + 0.5*radius*n);
				Coord target_j = (center - 0.5*radius*n);
//				pos_num += (center + 0.4*radius*n);

				atomicAdd(&newPoints[pId][0], target_i[0]);
				atomicAdd(&newPoints[j][0], target_j[0]);

				atomicAdd(&weights[pId], Real(1));
				atomicAdd(&weights[j], Real(1));

				if (Coord::dims() >= 2)
				{
					atomicAdd(&newPoints[pId][1], target_i[1]);
					atomicAdd(&newPoints[j][1], target_j[1]);
				}

				if (Coord::dims() >= 3)
				{
					atomicAdd(&newPoints[pId][2], target_i[2]);
					atomicAdd(&newPoints[j][2], target_j[2]);
				}
			}
		}

//		if (col_num != 0)
//			pos_num /= col_num;
//		else
//			pos_num = pos_i;
//
//		newPoints[pId] = pos_num;
	}

	template<typename Real, typename Coord>
	__global__ void K_ComputeTarget(
		GArray<Coord> oldPoints,
		GArray<Coord> newPoints, 
		GArray<Real> weights)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= oldPoints.size()) return;

		if (weights[pId] > EPSILON)
		{
			newPoints[pId] /= weights[pId];
		}
		else
			newPoints[pId] = oldPoints[pId];
	}

	template<typename Real, typename Coord>
	__global__ void K_ComputeVelocity(
		GArray<Coord> initPoints,
		GArray<Coord> curPoints,
		GArray<Coord> velocites,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= velocites.size()) return;

		velocites[pId] += (curPoints[pId] - initPoints[pId]) / dt;
	}

	template<typename TDataType>
	void RodCollision<TDataType>::doCollision()
	{
		int start = 0;
		for (int i = 0; i < m_collidableObjects.size(); i++)
		{
			GArray<Coord>& points = m_collidableObjects[i]->getPositions();
			GArray<Coord>& vels = m_collidableObjects[i]->getVelocities();
			int num = points.size();
			hipMemcpy(m_points.begin() + start, points.begin(), num * sizeof(Coord), hipMemcpyDeviceToDevice);
			hipMemcpy(m_vels.begin() + start, vels.begin(), num * sizeof(Coord), hipMemcpyDeviceToDevice);
			start += num;
		}

		if (m_nbrQuery == nullptr)
		{
			m_nbrQuery = std::make_shared<NeighborQuery<TDataType>>();
		}
		if (m_nList == nullptr)
		{
			m_nList = std::make_shared<NeighborList<int>>();
			m_nList->resize(m_points.size());
			m_nList->setNeighborLimit(5);
		}
		

		Real radius = 0.005;
		m_nbrQuery->queryParticleNeighbors(*m_nList, m_points, radius);

		GArray<Coord> posBuf;
		posBuf.resize(m_points.size());

		GArray<Real> weights;
		weights.resize(m_points.size());

		GArray<Coord> init_pos;
		init_pos.resize(m_points.size());

		init_pos.assign(m_points);

		uint pDims = cudaGridSize(m_points.size(), BLOCK_SIZE);
		for (size_t it = 0; it < 5; it++)
		{
			weights.reset();
			posBuf.reset();
			K_Collide << <pDims, BLOCK_SIZE >> > (m_objId, m_points, posBuf, weights, *m_nList, radius);
			K_ComputeTarget << <pDims, BLOCK_SIZE >> > (m_points, posBuf, weights);
			m_points.assign(posBuf);
		}

		K_ComputeVelocity << <pDims, BLOCK_SIZE >> > (init_pos, m_points, m_vels, getParent()->getDt());

		posBuf.clear();
		weights.clear();
		init_pos.clear();

		start = 0;
		for (int i = 0; i < m_collidableObjects.size(); i++)
		{
			GArray<Coord>& points = m_collidableObjects[i]->getPositions();
			GArray<Coord>& vels = m_collidableObjects[i]->getVelocities();
			int num = points.size();
			hipMemcpy(points.begin(), m_points.begin() + start, num * sizeof(Coord), hipMemcpyDeviceToDevice);
			hipMemcpy(vels.begin(), m_vels.begin() + start, num * sizeof(Coord), hipMemcpyDeviceToDevice);

			m_collidableObjects[i]->updateMechanicalState();
			start += num;
		}
	}


	template<typename TDataType>
	bool RodCollision<TDataType>::initializeImpl()
	{
		for (int i = 0; i < m_collidableObjects.size(); i++)
		{
			m_collidableObjects[i]->initialize();
		}

		size_t totalNum = 0;
		std::vector<int> ids;
		std::vector<Coord> hPos;
		for (int i = 0; i < m_collidableObjects.size(); i++)
		{
			GArray<Coord>& points = m_collidableObjects[i]->getPositions();
			for (int j = 0; j < points.size(); j++)
			{
				ids.push_back(i);
			}
			totalNum += points.size();
		}

		if (totalNum <= 0)
			return false;

		m_objId.resize(totalNum);
		m_points.resize(totalNum);
		m_vels.resize(totalNum);

		m_objId.assign(ids);

		return true;
	}

}