#include "hip/hip_runtime.h"
#include "RigidBodySystem.h"

namespace dyno
{
	IMPLEMENT_CLASS_1(RigidBodySystem, TDataType)

	typedef typename TOrientedBox3D<Real> Box3D;

	template<typename TDataType>
	RigidBodySystem<TDataType>::RigidBodySystem(std::string name)
		: Node(name)
	{
		auto defaultTopo = std::make_shared<DiscreteElements<TDataType>>();
		this->currentTopology()->setDataPtr(std::make_shared<DiscreteElements<TDataType>>());

		mElementQuery = std::make_shared<NeighborElementQuery<TDataType>>();
		this->currentTopology()->connect(mElementQuery->inDiscreteElements());
	}

	template<typename TDataType>
	RigidBodySystem<TDataType>::~RigidBodySystem()
	{
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::addBox(
		const BoxInfo& box,
		const RigidBodyInfo& bodyDef, 
		const Real density)
	{
		auto b = box;
		auto bd = bodyDef;

		float lx = 2.0f * b.halfLength[0];
		float ly = 2.0f * b.halfLength[1];
		float lz = 2.0f * b.halfLength[2];
		bd.position = b.center;

		bd.mass = density * lx * ly * lz;
		bd.inertia = 1.0f / 12.0f * bd.mass
			* Mat3f(ly*ly + lz * lz, 0, 0,
				0, lx*lx + lz * lz, 0,
				0, 0, lx*lx + ly * ly);

		bd.shapeType = ST_Box;
		bd.angle = b.rot;

		mHostRigidBodyStates.insert(mHostRigidBodyStates.begin() + mHostSpheres.size() + mHostBoxes.size(), bd);
		mHostBoxes.push_back(b);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::addSphere(
		const SphereInfo& sphere, 
		const RigidBodyInfo& bodyDef,
		const Real density /*= Real(1)*/)
	{
		auto b = sphere;
		auto bd = bodyDef;

		bd.position = b.center;

		float r = b.radius;
		if (bd.mass <= 0.0f) {
			bd.mass = 3 / 4.0f*M_PI*r*r*r*density;
		}
		float I11 = r * r;
		bd.inertia = 0.4f * bd.mass
			* Mat3f(I11, 0, 0,
				0, I11, 0,
				0, 0, I11);

		bd.shapeType = ST_Sphere;
		bd.angle = b.rot;

		mHostRigidBodyStates.insert(mHostRigidBodyStates.begin() + mHostSpheres.size(), bd);
		mHostSpheres.push_back(b);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::addTet(
		const TetInfo& tet,
		const RigidBodyInfo& bodyDef, 
		const Real density /*= Real(1)*/)
	{
		auto b = tet;
		auto bd = bodyDef;

		bd.position = (tet.v[0] + tet.v[1] + tet.v[2] + tet.v[3]) / 4;

		float r = 0.025;
		if (bd.mass <= 0.0f) {
			bd.mass = 3 / 4.0f*M_PI*r*r*r*density;
		}
		float I11 = r * r;
		bd.inertia = 0.4f * bd.mass
			* Mat3f(I11, 0, 0,
				0, I11, 0,
				0, 0, I11);

		bd.shapeType = ST_Tet;
		bd.angle = Quat<Real>();

		mHostRigidBodyStates.insert(mHostRigidBodyStates.begin() + mHostSpheres.size() + mHostBoxes.size() + mHostTets.size(), bd);
		mHostTets.push_back(b);
	}

	template <typename Real, typename Coord, typename Matrix, typename Quat>
	__global__ void RB_SetupInitialStates(
		DArray<Real> mass,
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		DArray<Coord> velocity,
		DArray<Coord> angularVelocity,
		DArray<Quat> rotation_q,
		DArray<Matrix> inertia,
		DArray<RigidBodyInfo> states,
		ElementOffset offset,
		int start_mesh)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= start_mesh) return;

		if (tId >= rotation_q.size())
			return;
		
		mass[tId] = states[tId].mass;
		rotation[tId] = states[tId].angle.toMatrix3x3();
		velocity[tId] = states[tId].linearVelocity;
		angularVelocity[tId] = states[tId].angularVelocity;
		rotation_q[tId] = states[tId].angle;
		pos[tId] = states[tId].position;
		inertia[tId] = states[tId].inertia;

// 		if (tId >= offset.segOffset) {}
// 		else if (tId >= offset.tetOffset) pos[tId] = (tets[tId - offset.tetOffset].v[0] + tets[tId - offset.tetOffset].v[1] + tets[tId - offset.tetOffset].v[2] + tets[tId - offset.tetOffset].v[3]) / 4.0f;
// 		else if (tId >= offset.boxOffset) pos[tId] = boxes[tId - offset.boxOffset].center;
// 		else pos[tId] = spheres[tId].center;
	}

	__global__ void SetupBoxes(
		DArray<Box3D> box3d,
		DArray<BoxInfo> boxInfo)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boxInfo.size()) return;

		box3d[tId].center = boxInfo[tId].center;
		box3d[tId].extent = boxInfo[tId].halfLength;

		Mat3f rot = boxInfo[tId].rot.toMatrix3x3();

		box3d[tId].u = rot * Vec3f(1, 0, 0);
		box3d[tId].v = rot * Vec3f(0, 1, 0);
		box3d[tId].w = rot * Vec3f(0, 0, 1);
	}

	__global__ void SetupSpheres(
		DArray<Sphere3D> sphere3d,
		DArray<SphereInfo> sphereInfo)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInfo.size()) return;

		sphere3d[tId].radius = sphereInfo[tId].radius;
		sphere3d[tId].center = sphereInfo[tId].center;
	}

	__global__ void SetupTets(
		DArray<Tet3D> tet3d,
		DArray<TetInfo> tetInfo)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tetInfo.size()) return;

		tet3d[tId].v[0] = tetInfo[tId].v[0];
		tet3d[tId].v[1] = tetInfo[tId].v[1];
		tet3d[tId].v[2] = tetInfo[tId].v[2];
		tet3d[tId].v[3] = tetInfo[tId].v[3];
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::resetStates()
	{
		auto topo = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());

		mDeviceBoxes.assign(mHostBoxes);
		mDeviceSpheres.assign(mHostSpheres);
		mDeviceTets.assign(mHostTets);

		auto& boxes = topo->getBoxes();
		auto& spheres = topo->getSpheres();
		auto& tets = topo->getTets();

		boxes.resize(mDeviceBoxes.size());
		spheres.resize(mDeviceSpheres.size());
		tets.resize(mDeviceTets.size());

		//Setup the topology
		cuExecute(mDeviceBoxes.size(),
			SetupBoxes,
			boxes,
			mDeviceBoxes);

		cuExecute(mDeviceSpheres.size(),
			SetupSpheres,
			spheres,
			mDeviceSpheres);

		cuExecute(mDeviceTets.size(),
			SetupTets,
			tets,
			mDeviceTets);

		mDeviceRigidBodyStates.assign(mHostRigidBodyStates);

		int sizeOfRigids = topo->totalSize();

		ElementOffset eleOffset = topo->calculateElementOffset();

		this->currentRotationMatrix()->setElementCount(sizeOfRigids);
		this->currentAngularVelocity()->setElementCount(sizeOfRigids);
		this->currentCenter()->setElementCount(sizeOfRigids);
		this->currentVelocity()->setElementCount(sizeOfRigids);
		this->currentMass()->setElementCount(sizeOfRigids);
		this->currentInertia()->setElementCount(sizeOfRigids);
		this->currentQuaternion()->setElementCount(sizeOfRigids);

		mBoundaryContactCounter.resize(sizeOfRigids);

		cuExecute(sizeOfRigids,
			RB_SetupInitialStates,
			this->currentMass()->getData(),
			this->currentCenter()->getData(),
			this->currentRotationMatrix()->getData(),
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			this->currentQuaternion()->getData(),
			this->currentInertia()->getData(),
			mDeviceRigidBodyStates,
			eleOffset,
			sizeOfRigids);

		mInitialInertia.assign(this->currentInertia()->getData());
	}
	
	template <typename Coord>
	__global__ void UpdateSpheres(
		DArray<Sphere3D> sphere,
		DArray<Coord> pos,
		int start_sphere)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= sphere.size()) return;

		sphere[pId].center = pos[pId + start_sphere];
	}

	template <typename Coord, typename Matrix>
	__global__ void UpdateBoxes(
		DArray<Box3D> box,
		DArray<BoxInfo> box_init,
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		int start_box)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= box.size()) return;
		box[pId].center = pos[pId + start_box];

		box[pId].extent = box_init[pId].halfLength;

		box[pId].u = rotation[pId + start_box] * Coord(1, 0, 0);
		box[pId].v = rotation[pId + start_box] * Coord(0, 1, 0);
		box[pId].w = rotation[pId + start_box] * Coord(0, 0, 1);
	}

	template <typename Coord, typename Matrix>
	__global__ void UpdateTets(
		DArray<Tet3D> tet,
		DArray<TetInfo> tet_init,
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		int start_tet)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= tet.size()) return;

		Coord3D center_init = (tet_init[pId].v[0] + tet_init[pId].v[1] + tet_init[pId].v[2] + tet_init[pId].v[3]) / 4.0f;
		tet[pId].v[0] = rotation[pId + start_tet] * (tet_init[pId].v[0] - center_init) + pos[pId + start_tet];
		tet[pId].v[1] = rotation[pId + start_tet] * (tet_init[pId].v[1] - center_init) + pos[pId + start_tet];
		tet[pId].v[2] = rotation[pId + start_tet] * (tet_init[pId].v[2] - center_init) + pos[pId + start_tet];
		tet[pId].v[3] = rotation[pId + start_tet] * (tet_init[pId].v[3] - center_init) + pos[pId + start_tet];
	}

	template <typename Coord, typename Matrix, typename Quat>
	__global__ void RB_UpdateGesture(
		DArray<Coord> pos,
		DArray<Quat> rotQuat,
		DArray<Matrix> rotMat,
		DArray<Matrix> inertia,
		DArray<Coord> velocity,
		DArray<Coord> angular_velocity,
		DArray<Matrix> inertia_init,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;
		
		pos[pId] += velocity[pId] * dt;

		rotQuat[pId] = rotQuat[pId].normalize();
		rotMat[pId] = rotQuat[pId].toMatrix3x3();
		
		rotQuat[pId] += dt * 0.5f * 
			Quat(angular_velocity[pId][0], angular_velocity[pId][1],angular_velocity[pId][2], 0.0)
			*(rotQuat[pId]);

		inertia[pId] = rotMat[pId] * inertia_init[pId] * rotMat[pId].inverse();
	}

	template <typename Coord>
	__global__ void RB_UpdateVelocity(
		DArray<Coord> velocity,
		DArray<Coord> angular_velocity,
		DArray<Coord> accel,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= accel.size() / 2) return;

		velocity[pId] += accel[2 * pId] * dt;
		velocity[pId] += Coord(0, -9.8f, 0) * dt;

		angular_velocity[pId] += accel[2 * pId + 1] * dt;
	}

	template <typename Coord, typename Matrix, typename ContactPair>
	__global__ void CalculateJacobians(
		DArray<Coord> J,
		DArray<Coord> B,
		DArray<Coord> pos,
		DArray<Matrix> inertia,
		DArray<Real> mass,
		DArray<ContactPair> nbc)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbc[pId].bodyId1;
		int idx2 = nbc[pId].bodyId2;

		//printf("%d %d\n", idx1, idx2);

		if (nbc[pId].contactType == ContactType::CT_NONPENETRATION) // contact, collision
		{
			Coord p1 = nbc[pId].pos1;
			Coord p2 = nbc[pId].pos2;
			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = -n;
			J[4 * pId + 3] = -(r2.cross(n));

			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			B[4 * pId + 2] = -n / mass[idx2];
			B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n));
		}
		else if (nbc[pId].contactType == ContactType::CT_BOUDNARY) // boundary
		{
			Coord p1 = nbc[pId].pos1;
		//	printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@ %d %.3lf %.3lf %.3lf\n", idx1, p1[0], p1[1], p1[2]);

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = Coord(0);
			J[4 * pId + 3] = Coord(0);

			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			B[4 * pId + 2] = Coord(0);
			B[4 * pId + 3] = Coord(0);
		}
		else if (nbc[pId].contactType == ContactType::CT_FRICTION) // friction
		{
			Coord p1 = nbc[pId].pos1;
			//printf("~~~~~~~ %.3lf %.3lf %.3lf\n", p1[0], p1[1], p1[2]);

			
			Coord p2 = Coord(0);
			if(idx2 != -1)
				p2 = nbc[pId].pos2;

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = Coord(0);
			if (idx2 != -1)
				r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			if(idx2 != -1)
			{ 
				J[4 * pId + 2] = -n;
				J[4 * pId + 3] = -(r2.cross(n));
			}
			else
			{
				J[4 * pId + 2] = Coord(0);
				J[4 * pId + 3] = Coord(0);
			}
			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			if(idx2 != -1)
			{ 
				B[4 * pId + 2] = -n / mass[idx2];
				B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n));
			}
			else
			{
				B[4 * pId + 2] = Coord(0);
				B[4 * pId + 3] = Coord(0);
			}
		}
	}

	template <typename Coord, typename Matrix, typename ContactPair>
	__global__ void CalculateJacobians(
		DArray<Coord> J,
		DArray<Coord> B,
		DArray<Coord> pos,
		DArray<Matrix> inertia,
		DArray<Matrix> inertia_eq,
		DArray<Real> mass,
		DArray<Real> mass_eq,
		DArray<ContactPair> nbc)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbc[pId].bodyId1;
		int idx2 = nbc[pId].bodyId2;

		//printf("%d %d\n", idx1, idx2);
		//EPSILON

		if (nbc[pId].contactType == ContactType::CT_NONPENETRATION) // contact, collision
		{
			Coord p1 = nbc[pId].pos1;
			Coord p2 = nbc[pId].pos2;
			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = -n;
			J[4 * pId + 3] = -(r2.cross(n));

			n /= n.norm();
			Coord3D n1(0), n2(0);

			Real ratio1 = 1.0f;
			Real ratio2 = 1.0f;

			if (n[0] > EPSILON)
			{
				n1 += Coord(n[0], 0, 0) / mass_eq[6 * idx1 + 1];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 1];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 1]);
				n2 -= Coord(n[0], 0, 0) / mass_eq[6 * idx2 + 0];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 0];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 0]);

			}
			else
			{
				n1 += Coord(n[0], 0, 0) / mass_eq[6 * idx1 + 0];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 0];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 0]);
				n2 -= Coord(n[0], 0, 0) / mass_eq[6 * idx2 + 1];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 1];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 1]);
			}
			if (n[1] > EPSILON)
			{
				n1 += Coord(0, n[1], 0) / mass_eq[6 * idx1 + 3];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 3];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 3]);
				n2 -= Coord(0, n[1], 0) / mass_eq[6 * idx2 + 2];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 2];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 2]);
			}
			else
			{
				n1 += Coord(0, n[1], 0) / mass_eq[6 * idx1 + 2];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 2];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 2]);
				n2 -= Coord(0, n[1], 0) / mass_eq[6 * idx2 + 3];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 3];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 3]);
			}
			if (n[2] > EPSILON)
			{
				n1 += Coord(0, 0, n[2]) / mass_eq[6 * idx1 + 5];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 5];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 5]);
				n2 -= Coord(0, 0, n[2]) / mass_eq[6 * idx2 + 4];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 4];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 4]);
			}
			else
			{
				n1 += Coord(0, 0, n[2]) / mass_eq[6 * idx1 + 4];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 4];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 4]);
				n2 -= Coord(0, 0, n[2]) / mass_eq[6 * idx2 + 5];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 5];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 5]);
			}

			ratio1 /= 3.0f;
			ratio2 /= 3.0f;

			printf("%.5lf %.5lf %.5lf %.5lf\n", ratio1, ratio2, mass[idx1], mass[idx2]);

			B[4 * pId] = n1;//n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n)) * ratio1;
			B[4 * pId + 2] = n2;//-n / mass[idx2];
			B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n)) * ratio2;
		}

		else if (nbc[pId].contactType == ContactType::CT_BOUDNARY) // boundary
		{

			Coord p1 = nbc[pId].pos1;
			//	printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@ %d %.3lf %.3lf %.3lf\n", idx1, p1[0], p1[1], p1[2]);

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];


			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = Coord(0);
			J[4 * pId + 3] = Coord(0);

			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			B[4 * pId + 2] = Coord(0);
			B[4 * pId + 3] = Coord(0);
		}

		else if (nbc[pId].contactType == ContactType::CT_FRICTION) // friction
		{
			Coord p1 = nbc[pId].pos1;
			//printf("~~~~~~~ %.3lf %.3lf %.3lf\n", p1[0], p1[1], p1[2]);


			Coord p2 = Coord(0);
			if (idx2 != -1)
				p2 = nbc[pId].pos2;

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = Coord(0);
			if (idx2 != -1)
				r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			if (idx2 != -1)
			{
				J[4 * pId + 2] = -n;
				J[4 * pId + 3] = -(r2.cross(n));
			}
			else
			{
				J[4 * pId + 2] = Coord(0);
				J[4 * pId + 3] = Coord(0);
			}
			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			if (idx2 != -1)
			{
				B[4 * pId + 2] = -n / mass[idx2];
				B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n));
			}
			else
			{
				B[4 * pId + 2] = Coord(0);
				B[4 * pId + 3] = Coord(0);
			}
		}

	}

// 	template <typename Coord, typename Matrix, typename Real>
// 	__global__ void RB_constrct_mass_eq(
// 		DArray<Coord> pos,
// 		DArray<Matrix> inertia,
// 		DArray<Real> mass,
// 		DArray<Coord> J,
// 		DArray<Coord> B,
// 		DArray<Real> mass_eq,
// 		DArray<Real> mass_eq_old,
// 		DArray<Matrix> inertia_eq,
// 		DArray<NeighborConstraints> nbc
// 	)
// 	{
// 		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
// 		if (pId >= J.size() / 4) return;
// 		int idx1 = nbc[pId].idx1;
// 		int idx2 = nbc[pId].idx2;
// 		if(nbc[pId].contactType != ContactType::CT_FRICTION)
// 		{
// 			Coord d = nbc[pId].normal1;
// 
// 			if (d[0] > EPSILON)
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if(idx2 != -1)
// 				{ 
// 					atomicAdd(&mass_eq[idx1 * 6], mass_eq_old[idx2 * 6] * d_n[0]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 1], mass_eq_old[idx1 * 6 + 1] * d_n[0]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6], 100000.0f);
// 				}
// 			}
// 			else
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 1], - mass_eq_old[idx2 * 6 + 1] * d_n[0]);
// 					atomicAdd(&mass_eq[idx2 * 6], - mass_eq_old[idx1 * 6] * d_n[0]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6], 100000.0f);
// 				}
// 			}
// 			if (d[1] > EPSILON)
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 2], mass_eq_old[idx2 * 6 + 2] * d_n[1]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 3], mass_eq_old[idx1 * 6 + 3] * d_n[1]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 2], 100000.0f);
// 				}
// 			}
// 			else
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 3], - mass_eq_old[idx2 * 6 + 3] * d_n[1]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 2], - mass_eq_old[idx1 * 6 + 2] * d_n[1]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 3], 100000.0f);
// 				}
// 			}
// 			if (d[2] > EPSILON)
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 4], mass_eq_old[idx2 * 6 + 4] * d_n[2]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 5], mass_eq_old[idx1 * 6 + 5] * d_n[2]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 4], 100000.0f);
// 				}
// 			}
// 			else
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 5], - mass_eq_old[idx2 * 6 + 5] * d_n[2]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 4], - mass_eq_old[idx1 * 6 + 4] * d_n[2]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 5], 100000.0f);
// 				}
// 			}
// 
// 		}
// 	}

// 	template <typename Real, typename Matrix>
// 	__global__ void RB_constrct_mass_eq(
// 		DArray<Matrix> inertia,
// 		DArray<Real> mass,
// 		DArray<Real> mass_eq
// 	)
// 	{
// 		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
// 		if (pId >= mass.size()) return;
// 
// 		for (int i = 0; i < 6; i++)
// 			mass_eq[pId * 6 + i] += mass[pId];
// 	}


	// ignore zeta !!!!!!
	template <typename Coord, typename ContactPair>
	__global__ void CalculateEta(
		DArray<Real> eta,
		DArray<Coord> velocity,
		DArray<Coord> angular_velocity,
		DArray<Coord> J,
		DArray<Real> mass,
		DArray<ContactPair> nbq,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbq[pId].bodyId1;
		int idx2 = nbq[pId].bodyId2;
		//printf("from ita %d\n", pId);
		Real ita_i = Real(0);
		if (true) // test dist constraint
		{
			ita_i -= J[4 * pId].dot(velocity[idx1]);
			ita_i -= J[4 * pId + 1].dot(angular_velocity[idx1]);
			if(idx2 != -1)
			{ 
				ita_i -= J[4 * pId + 2].dot(velocity[idx2]);
				ita_i -= J[4 * pId + 3].dot(angular_velocity[idx2]);
			}
		}
		eta[pId] = ita_i / dt;
		if (nbq[pId].contactType == ContactType::CT_NONPENETRATION || nbq[pId].contactType == ContactType::CT_BOUDNARY)
		{
			eta[pId] += min(nbq[pId].interpenetration, nbq[pId].interpenetration) / dt / dt / 15.0f;
		}

	}

	template <typename Coord>
	__global__ void CalculateDiagonals(
		DArray<Real> D,
		DArray<Coord> J,
		DArray<Coord> B)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= J.size() / 4) return;

		Real d = Real(0);
		d += J[4 * tId].dot(B[4 * tId]);
		d += J[4 * tId + 1].dot(B[4 * tId + 1]);
		d += J[4 * tId + 2].dot(B[4 * tId + 2]);
		d += J[4 * tId + 3].dot(B[4 * tId + 3]);

		D[tId] = d;
	}

	template <typename Coord, typename ContactPair>
	__global__ void TakeOneJacobiIteration(
		DArray<Real> lambda,
		DArray<Coord> accel,
		DArray<Real> d,
		DArray<Coord> J,
		DArray<Coord> B,
		DArray<Real> eta,
		DArray<Real> mass,
		DArray<ContactPair> nbq)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbq[pId].bodyId1;
		int idx2 = nbq[pId].bodyId2;

		Real eta_i = eta[pId];
		{
			eta_i -= J[4 * pId].dot(accel[idx1 * 2]);
			eta_i -= J[4 * pId + 1].dot(accel[idx1 * 2 + 1]);
			if (idx2 != -1)
			{
				eta_i -= J[4 * pId + 2].dot(accel[idx2 * 2]);
				eta_i -= J[4 * pId + 3].dot(accel[idx2 * 2 + 1]);
			}
		}

		if (d[pId] > EPSILON)
		{
			Real delta_lambda = eta_i / d[pId];
			delta_lambda *= 0.2;

			//printf("delta_lambda = %.3lf\n", delta_lambda);

			if (nbq[pId].contactType == ContactType::CT_NONPENETRATION || nbq[pId].contactType == ContactType::CT_BOUDNARY) //	PROJECTION!!!!
			{
				Real lambda_new = lambda[pId] + delta_lambda;
				if (lambda_new < 0) lambda_new = 0;

				Real mass_i = mass[idx1];
				if (idx2 != -1)
					mass_i += mass[idx2];

				if (lambda_new > 25 * (mass_i / 0.1)) lambda_new = 25 * (mass_i / 0.1);
				delta_lambda = lambda_new - lambda[pId];
			}

			if (nbq[pId].contactType == ContactType::CT_FRICTION) //	PROJECTION!!!!
			{
				Real lambda_new = lambda[pId] + delta_lambda;
				Real mass_i = mass[idx1];
				if (idx2 != -1)
					mass_i += mass[idx2];

				//if ((lambda_new) > 15 * (mass_i)) lambda_new = 15 * (mass_i);
				//if ((lambda_new) < -15 * (mass_i)) lambda_new = -15 * (mass_i);
				delta_lambda = lambda_new - lambda[pId];
			}

			lambda[pId] += delta_lambda;

			//printf("inside iteration: %d %d %.5lf   %.5lf\n", idx1, idx2, nbq[pId].s4, delta_lambda);

			atomicAdd(&accel[idx1 * 2][0], B[4 * pId][0] * delta_lambda);
			atomicAdd(&accel[idx1 * 2][1], B[4 * pId][1] * delta_lambda);
			atomicAdd(&accel[idx1 * 2][2], B[4 * pId][2] * delta_lambda);

			atomicAdd(&accel[idx1 * 2 + 1][0], B[4 * pId + 1][0] * delta_lambda);
			atomicAdd(&accel[idx1 * 2 + 1][1], B[4 * pId + 1][1] * delta_lambda);
			atomicAdd(&accel[idx1 * 2 + 1][2], B[4 * pId + 1][2] * delta_lambda);

			if (idx2 != -1)
			{
				atomicAdd(&accel[idx2 * 2][0], B[4 * pId + 2][0] * delta_lambda);
				atomicAdd(&accel[idx2 * 2][1], B[4 * pId + 2][1] * delta_lambda);
				atomicAdd(&accel[idx2 * 2][2], B[4 * pId + 2][2] * delta_lambda);

				atomicAdd(&accel[idx2 * 2 + 1][0], B[4 * pId + 3][0] * delta_lambda);
				atomicAdd(&accel[idx2 * 2 + 1][1], B[4 * pId + 3][1] * delta_lambda);
				atomicAdd(&accel[idx2 * 2 + 1][2], B[4 * pId + 3][2] * delta_lambda);
			}
		}

	}

	template<typename ContactPair>
	__global__ void RB_update_offset(
		DArray<ContactPair> nbq,
		int offset)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= nbq.size()) return;
		if(nbq[pId].bodyId1 != -1)
			nbq[pId].bodyId1 += offset;
		if(nbq[pId].bodyId2 != -1)
			nbq[pId].bodyId2 += offset;
	}

	template <typename Coord, typename ContactPair>
	__global__ void SetupContactsWithBoundary(
		DArray<Sphere3D> sphere,
		DArray<Box3D> box,
		DArray<Tet3D> tet,
		DArray<int> count,
		DArray<ContactPair> nbq,
		Coord hi,
		Coord lo,
		int start_sphere,
		int start_box,
		int start_tet,
		int start_segment)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= sphere.size() + box.size()) return;
		
		if (pId < start_box && pId >= start_sphere)//sphere
		{

		}
		else if (pId >= start_box && pId < start_tet)//box
		{
			//int idx = pId - start_box;
			int cnt = 0;
			int start_i = count[pId];
			Coord center = box[pId - start_box].center;
			Coord u = box[pId - start_box].u;
			Coord v = box[pId - start_box].v;
			Coord w = box[pId - start_box].w;
			Coord extent = box[pId - start_box].extent;
			Point3D p[8];
			p[0] = Point3D(center - u * extent[0] - v * extent[1] - w * extent[2]);
			p[1] = Point3D(center - u * extent[0] - v * extent[1] + w * extent[2]);
			p[2] = Point3D(center - u * extent[0] + v * extent[1] - w * extent[2]);
			p[3] = Point3D(center - u * extent[0] + v * extent[1] + w * extent[2]);
			p[4] = Point3D(center + u * extent[0] - v * extent[1] - w * extent[2]);
			p[5] = Point3D(center + u * extent[0] - v * extent[1] + w * extent[2]);
			p[6] = Point3D(center + u * extent[0] + v * extent[1] - w * extent[2]);
			p[7] = Point3D(center + u * extent[0] + v * extent[1] + w * extent[2]);
			bool c1, c2, c3, c4, c5, c6;
			c1 = c2 = c3 = c4 = c5 = c6 = true;
			for (int i = 0; i < 8; i++)
			{
				Coord pos = p[i].origin;
				if (pos[0] > hi[0] && c1)
				{
					c1 = true;
					nbq[cnt + start_i].bodyId1 = pId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(-1,0,0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = pos[0] - hi[0];
					cnt++;
				}
				if (pos[1] > hi[1] && c2)
				{
					c2 = true;
					nbq[cnt + start_i].bodyId1 = pId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = pos[1] - hi[1];
					cnt++;
				}
				if (pos[2] > hi[2] && c3)
				{
					c3 = true;
					nbq[cnt + start_i].bodyId1 = pId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = pos[2] - hi[2];
					cnt++;
				}
				if (pos[0] < lo[0] && c4)
				{
					c4 = true;
					nbq[cnt + start_i].bodyId1 = pId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo[0] - pos[0];
					cnt++;
				}
				if (pos[1] < lo[1] && c5)
				{
					c5 = true;
					nbq[cnt + start_i].bodyId1 = pId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo[1] - pos[1];
					cnt++;
				}
				if (pos[2] < lo[2] && c6)
				{
					c6 = true;
					nbq[cnt + start_i].bodyId1 = pId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo[2] - pos[2];
					cnt++;
				}

			}

		}
		else if (pId >= start_tet && pId < start_segment) // tets
		{
		}
		else//segments 
		{}
	}

	template <typename ContactPair>
	__global__ void SetupFrictionConstraints(
		DArray<ContactPair> nbq,
		int contact_size)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= contact_size) return;

		Coord3D n = nbq[pId].normal1;
		n /= n.norm();

		Coord3D n1, n2;
		if (abs(n[1]) > EPSILON || abs(n[2]) > EPSILON)
		{
			n1 = Coord3D(0, n[2], -n[1]);
			n1 /= n1.norm();
			n2 = n1.cross(n);
			n2 /= n2.norm();
		}
		else if(abs(n[0]) > EPSILON)
		{
			n1 = Coord3D(n[2], 0, -n[0]);
			n1 /= n1.norm();
			n2 = n1.cross(n);
			n2 /= n2.norm();
		}

		nbq[pId * 2 + contact_size] = nbq[pId];
		nbq[pId * 2 + contact_size].contactType = ContactType::CT_FRICTION;
		nbq[pId * 2 + contact_size].normal1 = n1;
		nbq[pId * 2 + 1 + contact_size] = nbq[pId];
		nbq[pId * 2 + 1 + contact_size].contactType = ContactType::CT_FRICTION;
		nbq[pId * 2 + 1 + contact_size].normal1 = n2;
	}

	template <typename Coord>
	__global__ void CountContactsWithBoundary(
		DArray<Sphere3D> sphere,
		DArray<Box3D> box,
		DArray<Tet3D> tet,
		DArray<int> count,
		Coord hi,
		Coord lo,
		int start_sphere,
		int start_box,
		int start_tet,
		int start_segment)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= sphere.size() + box.size()) return;

		if (pId < start_box && pId >= start_sphere)//sphere
		{

		}
		else if (pId >= start_box && pId < start_tet)//box
		{
			//int idx = pId - start_box;
			int cnt = 0;
//				int start_i;
			Coord center = box[pId - start_box].center;
			Coord u = box[pId - start_box].u;
			Coord v = box[pId - start_box].v;
			Coord w = box[pId - start_box].w;
			Coord extent = box[pId - start_box].extent;
			Point3D p[8];
			p[0] = Point3D(center - u * extent[0] - v * extent[1] - w * extent[2]);
			p[1] = Point3D(center - u * extent[0] - v * extent[1] + w * extent[2]);
			p[2] = Point3D(center - u * extent[0] + v * extent[1] - w * extent[2]);
			p[3] = Point3D(center - u * extent[0] + v * extent[1] + w * extent[2]);
			p[4] = Point3D(center + u * extent[0] - v * extent[1] - w * extent[2]);
			p[5] = Point3D(center + u * extent[0] - v * extent[1] + w * extent[2]);
			p[6] = Point3D(center + u * extent[0] + v * extent[1] - w * extent[2]);
			p[7] = Point3D(center + u * extent[0] + v * extent[1] + w * extent[2]);
			bool c1, c2, c3, c4, c5, c6;
			c1 = c2 = c3 = c4 = c5 = c6 = true;
			for (int i = 0; i < 8; i++)
			{
				Coord pos = p[i].origin;
				if (pos[0] > hi[0] && c1)
				{
					c1 = true;
					cnt++;
				}
				if (pos[1] > hi[1] && c2)
				{
					c2 = true;
					cnt++;
				}
				if (pos[2] > hi[2] && c3)
				{
					c3 = true;
					cnt++;
				}
				if (pos[0] < lo[0] && c4)
				{
					c4 = true;
					cnt++;
				}
				if (pos[1] < lo[1] && c5)
				{
					c5 = true;
					cnt++;
				}
				if (pos[2] < lo[2] && c6)
				{
					c6 = true;
					cnt++;
				}
			}
			count[pId] = cnt;
		}
		else if (pId >= start_tet && pId < start_segment)//tets
		{
		}
		else//segments
		{}
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::detectCollisionWithBoundary()
	{
		auto discreteSet = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());
		uint totalSize = discreteSet->totalSize();

		ElementOffset offset = discreteSet->calculateElementOffset();

		int sum = 0;

		mBoundaryContactCounter.resize(discreteSet->totalSize());
		mBoundaryContactCounter.reset();
		if (discreteSet->totalSize() > 0)
		{
			cuExecute(totalSize,
				CountContactsWithBoundary,
				discreteSet->getSpheres(),
				discreteSet->getBoxes(),
				discreteSet->getTets(),
				mBoundaryContactCounter,
				mUpperCorner,
				mLowerCorner,
				0,
				offset.boxOffset,
				offset.tetOffset,
				offset.segOffset);

			sum += m_reduce.accumulate(mBoundaryContactCounter.begin(), mBoundaryContactCounter.size());
			m_scan.exclusive(mBoundaryContactCounter, true);

			mBoundaryContacts.resize(sum);

			if (sum > 0) {
				cuExecute(totalSize,
					SetupContactsWithBoundary,
					discreteSet->getSpheres(),
					discreteSet->getBoxes(),
					discreteSet->getTets(),
					mBoundaryContactCounter,
					mBoundaryContacts,
					mUpperCorner,
					mLowerCorner,
					0,
					offset.boxOffset,
					offset.tetOffset,
					offset.segOffset);
			}
		}
		else
			mBoundaryContacts.resize(0);

		if (have_mesh_boundary)
		{

			
		}
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::init_friction()
	{




	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::initializeJacobian(Real dt)
	{
		auto topo = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());

		mElementQuery->update();

		auto& contacts = mElementQuery->outContacts()->getData();

		detectCollisionWithBoundary();

		int sizeOfContacts = mBoundaryContacts.size();
		sizeOfContacts += contacts.size();

		int sizeOfConstraints = sizeOfContacts;
		if (this->varFrictionEnabled()->getData())
		{
			sizeOfConstraints += 2 * mBoundaryContacts.size();
			if (topo->totalSize() > 0)
				sizeOfConstraints += 2 * contacts.size();
		}

		mAllConstraints.resize(sizeOfConstraints);
		
		mJ.resize(4 * sizeOfConstraints);
		mB.resize(4 * sizeOfConstraints);
		mAccel.resize(currentCenter()->getElementCount() * 2);
		mD.resize(sizeOfConstraints);
		mEta.resize(sizeOfConstraints);
		mLambda.resize(sizeOfConstraints);

		mJ.reset();
		mB.reset();
		mD.reset();
		mEta.reset();
		mAccel.reset();
		mLambda.reset();

		if (sizeOfConstraints == 0) return;

		if (topo->totalSize() > 0 && contacts.size() > 0)
			mAllConstraints.assign(contacts, contacts.size());
		
		if (mBoundaryContacts.size() > 0)
		{
			if (topo->totalSize() > 0)
			{ 
				if (!have_mesh)
					mAllConstraints.assign(mBoundaryContacts, mBoundaryContacts.size(), contacts.size(), 0);
			}
			else
			{
				mAllConstraints.assign(mBoundaryContacts, mBoundaryContacts.size());
			}
		}

		if (this->varFrictionEnabled()->getData())
		{
			cuExecute(sizeOfContacts, 
				SetupFrictionConstraints,
				mAllConstraints,
				sizeOfContacts
				);
		}

		cuExecute(sizeOfConstraints,
			CalculateJacobians,
			mJ,
			mB,
			this->currentCenter()->getData(),
			this->currentInertia()->getData(),
			this->currentMass()->getData(),
			mAllConstraints);

		cuExecute(sizeOfConstraints,
			CalculateDiagonals,
			mD,
			mJ,
			mB);

		cuExecute(sizeOfConstraints, 
			CalculateEta,
			mEta,
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			mJ,
			this->currentMass()->getData(),
			mAllConstraints,
			dt);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::updateStates()
	{
		Real dt = this->varTimeStep()->getData();
		//construct j
		initializeJacobian(dt);
		for (int i = 0; i < 15; i++)
		{
			int size_constraints = mAllConstraints.size();
			if (size_constraints == 0) return;
			uint pDims = cudaGridSize(size_constraints, BLOCK_SIZE);

			TakeOneJacobiIteration << <pDims, BLOCK_SIZE >> > (
				mLambda,
				mAccel,
				mD,
				mJ,
				mB,
				mEta,
				currentMass()->getData(),
				mAllConstraints);
		}

		uint num = this->currentCenter()->getElementCount();
		cuExecute(num,
			RB_UpdateVelocity,
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			mAccel,
			dt);

		cuExecute(num,
			RB_UpdateGesture,
			this->currentCenter()->getData(),
			this->currentQuaternion()->getData(),
			this->currentRotationMatrix()->getData(),
			this->currentInertia()->getData(),
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			mInitialInertia,
			dt);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::updateTopology()
	{
		auto discreteSet = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());

		ElementOffset offset = discreteSet->calculateElementOffset();

		cuExecute(mDeviceBoxes.size(),
			UpdateBoxes,
			discreteSet->getBoxes(),
			mDeviceBoxes,
			this->currentCenter()->getData(),
			this->currentRotationMatrix()->getData(),
			offset.boxOffset);

		cuExecute(mDeviceBoxes.size(),
			UpdateSpheres,
			discreteSet->getSpheres(),
			this->currentCenter()->getData(),
			0);

		cuExecute(mDeviceTets.size(),
			UpdateTets,
			discreteSet->getTets(),
			mDeviceTets,
			this->currentCenter()->getData(),
			this->currentRotationMatrix()->getData(),
			offset.tetOffset);
	}

	DEFINE_CLASS(RigidBodySystem);
}