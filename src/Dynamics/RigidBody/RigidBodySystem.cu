#include "hip/hip_runtime.h"
#include "RigidBodySystem.h"

namespace dyno
{
	IMPLEMENT_CLASS_1(RigidBodySystem, TDataType)

	template<typename TDataType>
	RigidBodySystem<TDataType>::RigidBodySystem(std::string name)
		: Node(name)
	{
		auto defaultTopo = std::make_shared<DiscreteElements<TDataType>>();
		this->currentTopology()->setDataPtr(std::make_shared<DiscreteElements<TDataType>>());

		mElementQuery = std::make_shared<NeighborElementQuery<TDataType>>();
		this->currentTopology()->connect(mElementQuery->inDiscreteElements());
	}

	template<typename TDataType>
	RigidBodySystem<TDataType>::~RigidBodySystem()
	{
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::addBox(
		const BoxInfo& box,
		const RigidBodyInfo& bodyDef, 
		const Real density)
	{
		auto b = box;
		auto bd = bodyDef;

		float lx = 2.0f * b.halfLength[0];
		float ly = 2.0f * b.halfLength[1];
		float lz = 2.0f * b.halfLength[2];
		bd.position = b.center;

		bd.mass = density * lx * ly * lz;
		bd.inertia = 1.0f / 12.0f * bd.mass
			* Mat3f(ly*ly + lz * lz, 0, 0,
				0, lx*lx + lz * lz, 0,
				0, 0, lx*lx + ly * ly);

		bd.shapeType = ST_Box;
		bd.angle = b.rot;

		mHostRigidBodyStates.insert(mHostRigidBodyStates.begin() + mHostSpheres.size() + mHostBoxes.size(), bd);
		mHostBoxes.push_back(b);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::addSphere(
		const SphereInfo& sphere, 
		const RigidBodyInfo& bodyDef,
		const Real density /*= Real(1)*/)
	{
		auto b = sphere;
		auto bd = bodyDef;

		bd.position = b.center;

		float r = b.radius;
		if (bd.mass <= 0.0f) {
			bd.mass = 3 / 4.0f*M_PI*r*r*r*density;
		}
		float I11 = r * r;
		bd.inertia = 0.4f * bd.mass
			* Mat3f(I11, 0, 0,
				0, I11, 0,
				0, 0, I11);

		bd.shapeType = ST_Sphere;
		bd.angle = b.rot;

		mHostRigidBodyStates.insert(mHostRigidBodyStates.begin() + mHostSpheres.size(), bd);
		mHostSpheres.push_back(b);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::addTet(
		const TetInfo& tet, 
		const RigidBodyInfo& bodyDef, 
		const Real density /*= Real(1)*/)
	{
		auto b = tet;
		auto bd = bodyDef;

		bd.position = (tet.v[0] + tet.v[1] + tet.v[2] + tet.v[3]) / 4;

		float r = 0.025;
		if (bd.mass <= 0.0f) {
			bd.mass = 3 / 4.0f*M_PI*r*r*r*density;
		}
		float I11 = r * r;
		bd.inertia = 0.4f * bd.mass
			* Mat3f(I11, 0, 0,
				0, I11, 0,
				0, 0, I11);

		bd.shapeType = ST_Tet;
		bd.angle = Quat1f();

		mHostRigidBodyStates.insert(mHostRigidBodyStates.begin() + mHostSpheres.size() + mHostBoxes.size() + mHostTets.size(), bd);
		mHostTets.push_back(b);
	}

	template <typename Real, typename Coord, typename Matrix, typename Quat>
	__global__ void RB_initialize_device(
		DArray<Real> mass,
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		DArray<Coord> velocity,
		DArray<Coord> angularVelocity,
		DArray<Quat> rotation_q,
		DArray<Matrix> inertia,
		DArray<RigidBodyInfo> states,
		ElementOffset offset,
		int start_mesh)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= start_mesh) return;

		if (tId >= rotation_q.size())
			return;
		
		mass[tId] = states[tId].mass;
		rotation[tId] = states[tId].angle.toMatrix3x3();
		velocity[tId] = states[tId].linearVelocity;
		angularVelocity[tId] = states[tId].angularVelocity;
		rotation_q[tId] = states[tId].angle;
		pos[tId] = states[tId].position;
		inertia[tId] = states[tId].inertia;

// 		if (tId >= offset.segOffset) {}
// 		else if (tId >= offset.tetOffset) pos[tId] = (tets[tId - offset.tetOffset].v[0] + tets[tId - offset.tetOffset].v[1] + tets[tId - offset.tetOffset].v[2] + tets[tId - offset.tetOffset].v[3]) / 4.0f;
// 		else if (tId >= offset.boxOffset) pos[tId] = boxes[tId - offset.boxOffset].center;
// 		else pos[tId] = spheres[tId].center;
	}

	__global__ void SetupBoxes(
		DArray<Box3D> box3d,
		DArray<BoxInfo> boxInfo)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boxInfo.size()) return;

		box3d[tId].center = boxInfo[tId].center;
		box3d[tId].extent = boxInfo[tId].halfLength;

		Mat3f rot = boxInfo[tId].rot.toMatrix3x3();

		box3d[tId].u = rot * Vec3f(1, 0, 0);
		box3d[tId].v = rot * Vec3f(0, 1, 0);
		box3d[tId].w = rot * Vec3f(0, 0, 1);
	}

	__global__ void SetupSpheres(
		DArray<Sphere3D> sphere3d,
		DArray<SphereInfo> sphereInfo)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInfo.size()) return;

		sphere3d[tId].radius = sphereInfo[tId].radius;
		sphere3d[tId].center = sphereInfo[tId].center;
	}

	__global__ void SetupTets(
		DArray<Tet3D> tet3d,
		DArray<TetInfo> tetInfo)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tetInfo.size()) return;

		tet3d[tId].v[0] = tetInfo[tId].v[0];
		tet3d[tId].v[1] = tetInfo[tId].v[1];
		tet3d[tId].v[2] = tetInfo[tId].v[2];
		tet3d[tId].v[3] = tetInfo[tId].v[3];
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::resetStates()
	{
		auto topo = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());

		mDeviceBoxes.assign(mHostBoxes);
		mDeviceSpheres.assign(mHostSpheres);
		mDeviceTets.assign(mHostTets);

		auto& boxes = topo->getBoxes();
		auto& spheres = topo->getSpheres();
		auto& tets = topo->getTets();

		boxes.resize(mDeviceBoxes.size());
		spheres.resize(mDeviceSpheres.size());
		tets.resize(mDeviceTets.size());

		cuExecute(mDeviceBoxes.size(),
			SetupBoxes,
			boxes,
			mDeviceBoxes);

		cuExecute(mDeviceSpheres.size(),
			SetupSpheres,
			spheres,
			mDeviceSpheres);

		cuExecute(mDeviceTets.size(),
			SetupTets,
			tets,
			mDeviceTets);

		mDeviceRigidBodyStates.assign(mHostRigidBodyStates);

		int sizeOfRigids = topo->totalSize();

		ElementOffset eleOffset = topo->calculateElementOffset();

		this->currentRigidRotation()->setElementCount(sizeOfRigids);
		this->currentAngularVelocity()->setElementCount(sizeOfRigids);
		this->currentCenter()->setElementCount(sizeOfRigids);
		this->currentVelocity()->setElementCount(sizeOfRigids);
		this->currentMass()->setElementCount(sizeOfRigids);
		this->currentInertia()->setElementCount(sizeOfRigids);
		this->currentRotation()->setElementCount(sizeOfRigids);

		mBoundaryContactCounter.resize(sizeOfRigids);

//		mass_eq.resize(sizeOfRigids * 6);

		uint pDimsR = cudaGridSize(sizeOfRigids, BLOCK_SIZE);
		
		RB_initialize_device << <pDimsR, BLOCK_SIZE >> > (
			this->currentMass()->getData(),
			this->currentCenter()->getData(),
			this->currentRigidRotation()->getData(),
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			this->currentRotation()->getData(),
			this->currentInertia()->getData(),
			mDeviceRigidBodyStates,
			eleOffset,
			sizeOfRigids);
	
		center_init.resize(sizeOfRigids);
		center_init.assign(this->currentCenter()->getData());

		m_inertia_init.assign(this->currentInertia()->getData());
	}
	
	template <typename Coord>
	__global__ void UpdateSpheres(
		DArray<Sphere3D> sphere,
		DArray<Coord> pos,
		int start_sphere)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= sphere.size()) return;

		sphere[pId].center = pos[pId + start_sphere];
	}

	template <typename Coord, typename Matrix>
	__global__ void UpdateBoxes(
		DArray<Box3D> box,
		DArray<BoxInfo> box_init,
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		int start_box)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= box.size()) return;
		box[pId].center = pos[pId + start_box];

		box[pId].extent = box_init[pId].halfLength;

		box[pId].u = rotation[pId + start_box] * Coord(1, 0, 0);
		box[pId].v = rotation[pId + start_box] * Coord(0, 1, 0);
		box[pId].w = rotation[pId + start_box] * Coord(0, 0, 1);
	}

	template <typename Coord, typename Matrix>
	__global__ void UpdateTets(
		DArray<Tet3D> tet,
		DArray<TetInfo> tet_init,
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		int start_tet)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= tet.size()) return;

		Coord3D center_init = (tet_init[pId].v[0] + tet_init[pId].v[1] + tet_init[pId].v[2] + tet_init[pId].v[3]) / 4.0f;
		tet[pId].v[0] = rotation[pId + start_tet] * (tet_init[pId].v[0] - center_init) + pos[pId + start_tet];
		tet[pId].v[1] = rotation[pId + start_tet] * (tet_init[pId].v[1] - center_init) + pos[pId + start_tet];
		tet[pId].v[2] = rotation[pId + start_tet] * (tet_init[pId].v[2] - center_init) + pos[pId + start_tet];
		tet[pId].v[3] = rotation[pId + start_tet] * (tet_init[pId].v[3] - center_init) + pos[pId + start_tet];
	}

	template <typename Coord, typename Matrix, typename Quat>
	__global__ void RB_update_state(
		DArray<Coord> pos,
		DArray<Matrix> rotation,
		DArray<Quat> rotation_q,
		DArray<Coord> velocity,
		DArray<Coord> angular_velocity,
		DArray<Matrix> inertia,
		DArray<Matrix> inertia_init,
		Real dt
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;
		
		pos[pId] += velocity[pId] * dt;
		
		rotation_q[pId] += dt * 0.5f * 
			Quat(angular_velocity[pId][0], angular_velocity[pId][1],angular_velocity[pId][2], 0.0f)
			*
			(rotation_q[pId]);
		
		rotation_q[pId] = rotation_q[pId].normalize();
		rotation[pId] = rotation_q[pId].toMatrix3x3();

		inertia[pId] = rotation[pId] * inertia_init[pId] * rotation[pId].inverse();
	}

	template <typename Coord>
	__global__ void RB_update_velocity(
		DArray<Coord> velocity,
		DArray<Coord> angular_velocity,
		DArray<Coord> AA,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= AA.size() / 2) return;

		//printf("%.3lf %.3lf %.3lf\n", AA[2 * pId][0], AA[2 * pId][1], AA[2 * pId][2]);
		velocity[pId] += AA[2 * pId] * dt;// + Coord(0, -9.8f, 0) * dt;
		velocity[pId] += Coord(0, -9.8f, 0) * dt;
		 //printf("velocity: %.3lf %.3lf %.3lf\n", velocity[pId][0], velocity[pId][1], velocity[pId][2]);
		angular_velocity[pId] += AA[2 * pId + 1] * dt;
	}

	template <typename Coord, typename Matrix>
	__global__ void CalculateJacobians(
		DArray<Coord> J,
		DArray<Coord> B,
		DArray<Coord> pos,
		DArray<Matrix> inertia,
		DArray<Real> mass,
		DArray<NeighborConstraints> nbc)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbc[pId].idx1;
		int idx2 = nbc[pId].idx2;

		//printf("%d %d\n", idx1, idx2);

		if (nbc[pId].constraint_type == constraint_distance) // test dist constraint
		{
			Coord p2 = nbc[pId].pos2;
			Coord p1 = nbc[pId].pos1;
			Coord d = p2 - p1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = p2 - pos[idx2];

			J[4 * pId] = -d ;
			J[4 * pId + 1] = (-r1.cross(d));
			J[4 * pId + 2] = d ;
			J[4 * pId + 3] = (r2.cross(d));

			B[4 * pId] = -d / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (- r1.cross(d)) ;
			B[4 * pId + 2] = d / mass[idx2];
			B[4 * pId + 3] = inertia[idx2].inverse() * (r2.cross(d));
		}
		else if (nbc[pId].constraint_type == constraint_collision) // contact, collision
		{
			Coord p1 = nbc[pId].pos1;
			Coord p2 = nbc[pId].pos2;
			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = -n;
			J[4 * pId + 3] = -(r2.cross(n));

			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			B[4 * pId + 2] = -n / mass[idx2];
			B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n));
		}
		else if (nbc[pId].constraint_type == constraint_boundary) // boundary
		{
			Coord p1 = nbc[pId].pos1;
		//	printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@ %d %.3lf %.3lf %.3lf\n", idx1, p1[0], p1[1], p1[2]);

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = Coord(0);
			J[4 * pId + 3] = Coord(0);

			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			B[4 * pId + 2] = Coord(0);
			B[4 * pId + 3] = Coord(0);
		}
		else if (nbc[pId].constraint_type == constraint_friction) // friction
		{
			Coord p1 = nbc[pId].pos1;
			//printf("~~~~~~~ %.3lf %.3lf %.3lf\n", p1[0], p1[1], p1[2]);

			
			Coord p2 = Coord(0);
			if(idx2 != -1)
				p2 = nbc[pId].pos2;

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = Coord(0);
			if (idx2 != -1)
				r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			if(idx2 != -1)
			{ 
				J[4 * pId + 2] = -n;
				J[4 * pId + 3] = -(r2.cross(n));
			}
			else
			{
				J[4 * pId + 2] = Coord(0);
				J[4 * pId + 3] = Coord(0);
			}
			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			if(idx2 != -1)
			{ 
				B[4 * pId + 2] = -n / mass[idx2];
				B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n));
			}
			else
			{
				B[4 * pId + 2] = Coord(0);
				B[4 * pId + 3] = Coord(0);
			}
		}
	}

	template <typename Coord, typename Matrix>
	__global__ void CalculateJacobians(
		DArray<Coord> J,
		DArray<Coord> B,
		DArray<Coord> pos,
		DArray<Matrix> inertia,
		DArray<Matrix> inertia_eq,
		DArray<Real> mass,
		DArray<Real> mass_eq,
		DArray<NeighborConstraints> nbc)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbc[pId].idx1;
		int idx2 = nbc[pId].idx2;

		//printf("%d %d\n", idx1, idx2);
		//EPSILON

		if (nbc[pId].constraint_type == constraint_collision) // contact, collision
		{
			Coord p1 = nbc[pId].pos1;
			Coord p2 = nbc[pId].pos2;
			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = -n;
			J[4 * pId + 3] = -(r2.cross(n));

			n /= n.norm();
			Coord3D n1(0), n2(0);

			Real ratio1 = 1.0f;
			Real ratio2 = 1.0f;

			if (n[0] > EPSILON)
			{
				n1 += Coord(n[0], 0, 0) / mass_eq[6 * idx1 + 1];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 1];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 1]);
				n2 -= Coord(n[0], 0, 0) / mass_eq[6 * idx2 + 0];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 0];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 0]);

			}
			else
			{
				n1 += Coord(n[0], 0, 0) / mass_eq[6 * idx1 + 0];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 0];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 0]);
				n2 -= Coord(n[0], 0, 0) / mass_eq[6 * idx2 + 1];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 1];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 1]);
			}
			if (n[1] > EPSILON)
			{
				n1 += Coord(0, n[1], 0) / mass_eq[6 * idx1 + 3];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 3];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 3]);
				n2 -= Coord(0, n[1], 0) / mass_eq[6 * idx2 + 2];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 2];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 2]);
			}
			else
			{
				n1 += Coord(0, n[1], 0) / mass_eq[6 * idx1 + 2];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 2];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 2]);
				n2 -= Coord(0, n[1], 0) / mass_eq[6 * idx2 + 3];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 3];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 3]);
			}
			if (n[2] > EPSILON)
			{
				n1 += Coord(0, 0, n[2]) / mass_eq[6 * idx1 + 5];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 5];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 5]);
				n2 -= Coord(0, 0, n[2]) / mass_eq[6 * idx2 + 4];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 4];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 4]);
			}
			else
			{
				n1 += Coord(0, 0, n[2]) / mass_eq[6 * idx1 + 4];
				//ratio1 += mass[idx1] / mass_eq[6 * idx1 + 4];
				ratio1 = min(ratio1, mass[idx1] / mass_eq[6 * idx1 + 4]);
				n2 -= Coord(0, 0, n[2]) / mass_eq[6 * idx2 + 5];
				//ratio2 += mass[idx2] / mass_eq[6 * idx2 + 5];
				ratio2 = min(ratio2, mass[idx2] / mass_eq[6 * idx2 + 5]);
			}

			ratio1 /= 3.0f;
			ratio2 /= 3.0f;

			printf("%.5lf %.5lf %.5lf %.5lf\n", ratio1, ratio2, mass[idx1], mass[idx2]);

			B[4 * pId] = n1;//n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n)) * ratio1;
			B[4 * pId + 2] = n2;//-n / mass[idx2];
			B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n)) * ratio2;
		}

		else if (nbc[pId].constraint_type == constraint_boundary) // boundary
		{

			Coord p1 = nbc[pId].pos1;
			//	printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@ %d %.3lf %.3lf %.3lf\n", idx1, p1[0], p1[1], p1[2]);

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];


			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			J[4 * pId + 2] = Coord(0);
			J[4 * pId + 3] = Coord(0);

			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			B[4 * pId + 2] = Coord(0);
			B[4 * pId + 3] = Coord(0);
		}

		else if (nbc[pId].constraint_type == constraint_friction) // friction
		{
			Coord p1 = nbc[pId].pos1;
			//printf("~~~~~~~ %.3lf %.3lf %.3lf\n", p1[0], p1[1], p1[2]);


			Coord p2 = Coord(0);
			if (idx2 != -1)
				p2 = nbc[pId].pos2;

			Coord n = nbc[pId].normal1;
			Coord r1 = p1 - pos[idx1];
			Coord r2 = Coord(0);
			if (idx2 != -1)
				r2 = p2 - pos[idx2];

			J[4 * pId] = n;
			J[4 * pId + 1] = (r1.cross(n));
			if (idx2 != -1)
			{
				J[4 * pId + 2] = -n;
				J[4 * pId + 3] = -(r2.cross(n));
			}
			else
			{
				J[4 * pId + 2] = Coord(0);
				J[4 * pId + 3] = Coord(0);
			}
			B[4 * pId] = n / mass[idx1];
			B[4 * pId + 1] = inertia[idx1].inverse() * (r1.cross(n));
			if (idx2 != -1)
			{
				B[4 * pId + 2] = -n / mass[idx2];
				B[4 * pId + 3] = inertia[idx2].inverse() * (-r2.cross(n));
			}
			else
			{
				B[4 * pId + 2] = Coord(0);
				B[4 * pId + 3] = Coord(0);
			}
		}

	}

// 	template <typename Coord, typename Matrix, typename Real>
// 	__global__ void RB_constrct_mass_eq(
// 		DArray<Coord> pos,
// 		DArray<Matrix> inertia,
// 		DArray<Real> mass,
// 		DArray<Coord> J,
// 		DArray<Coord> B,
// 		DArray<Real> mass_eq,
// 		DArray<Real> mass_eq_old,
// 		DArray<Matrix> inertia_eq,
// 		DArray<NeighborConstraints> nbc
// 	)
// 	{
// 		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
// 		if (pId >= J.size() / 4) return;
// 		int idx1 = nbc[pId].idx1;
// 		int idx2 = nbc[pId].idx2;
// 		if(nbc[pId].constraint_type != constraint_friction)
// 		{
// 			Coord d = nbc[pId].normal1;
// 
// 			if (d[0] > EPSILON)
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if(idx2 != -1)
// 				{ 
// 					atomicAdd(&mass_eq[idx1 * 6], mass_eq_old[idx2 * 6] * d_n[0]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 1], mass_eq_old[idx1 * 6 + 1] * d_n[0]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6], 100000.0f);
// 				}
// 			}
// 			else
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 1], - mass_eq_old[idx2 * 6 + 1] * d_n[0]);
// 					atomicAdd(&mass_eq[idx2 * 6], - mass_eq_old[idx1 * 6] * d_n[0]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6], 100000.0f);
// 				}
// 			}
// 			if (d[1] > EPSILON)
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 2], mass_eq_old[idx2 * 6 + 2] * d_n[1]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 3], mass_eq_old[idx1 * 6 + 3] * d_n[1]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 2], 100000.0f);
// 				}
// 			}
// 			else
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 3], - mass_eq_old[idx2 * 6 + 3] * d_n[1]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 2], - mass_eq_old[idx1 * 6 + 2] * d_n[1]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 3], 100000.0f);
// 				}
// 			}
// 			if (d[2] > EPSILON)
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 4], mass_eq_old[idx2 * 6 + 4] * d_n[2]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 5], mass_eq_old[idx1 * 6 + 5] * d_n[2]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 4], 100000.0f);
// 				}
// 			}
// 			else
// 			{
// 				Coord3D d_n = d / d.norm();
// 				if (idx2 != -1)
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 5], - mass_eq_old[idx2 * 6 + 5] * d_n[2]);
// 					atomicAdd(&mass_eq[idx2 * 6 + 4], - mass_eq_old[idx1 * 6 + 4] * d_n[2]);
// 				}
// 				else
// 				{
// 					atomicAdd(&mass_eq[idx1 * 6 + 5], 100000.0f);
// 				}
// 			}
// 
// 		}
// 	}

// 	template <typename Real, typename Matrix>
// 	__global__ void RB_constrct_mass_eq(
// 		DArray<Matrix> inertia,
// 		DArray<Real> mass,
// 		DArray<Real> mass_eq
// 	)
// 	{
// 		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
// 		if (pId >= mass.size()) return;
// 
// 		for (int i = 0; i < 6; i++)
// 			mass_eq[pId * 6 + i] += mass[pId];
// 	}


	// ignore zeta !!!!!!
	template <typename Coord>
	__global__ void CalculateEta(
		DArray<Real> eta,
		DArray<Coord> velocity,
		DArray<Coord> angular_velocity,
		DArray<Coord> J,
		DArray<Real> mass,
		DArray<NeighborConstraints> nbq,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbq[pId].idx1;
		int idx2 = nbq[pId].idx2;
		//printf("from ita %d\n", pId);
		Real ita_i = Real(0);
		if (true) // test dist constraint
		{
			ita_i -= J[4 * pId].dot(velocity[idx1]);
			ita_i -= J[4 * pId + 1].dot(angular_velocity[idx1]);
			if(idx2 != -1)
			{ 
				ita_i -= J[4 * pId + 2].dot(velocity[idx2]);
				ita_i -= J[4 * pId + 3].dot(angular_velocity[idx2]);
			}
		}
		eta[pId] = ita_i / dt;
		if (nbq[pId].constraint_type == constraint_collision || nbq[pId].constraint_type == constraint_boundary)
		{
			eta[pId] += min(nbq[pId].inter_distance, nbq[pId].inter_distance) / dt / dt / 15.0f;
		}

	}

	template <typename Coord>
	__global__ void CalculateDiagonals(
		DArray<Real> D,
		DArray<Coord> J,
		DArray<Coord> B)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= J.size() / 4) return;

		Real d = Real(0);
		d += J[4 * tId].dot(B[4 * tId]);
		d += J[4 * tId + 1].dot(B[4 * tId + 1]);
		d += J[4 * tId + 2].dot(B[4 * tId + 2]);
		d += J[4 * tId + 3].dot(B[4 * tId + 3]);

		D[tId] = d;
	}

	template <typename Coord>
	__global__ void TakeOneJacobiIteration(
		DArray<Real> lambda,
		DArray<Coord> AA,
		DArray<Real> d,
		DArray<Coord> J,
		DArray<Coord> B,
		DArray<Real> ita,
		DArray<Real> mass,
		DArray<NeighborConstraints> nbq)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= J.size() / 4) return;

		int idx1 = nbq[pId].idx1;
		int idx2 = nbq[pId].idx2;

		Real ita_i = ita[pId];
		{
			ita_i -= J[4 * pId].dot(AA[idx1 * 2]);
			ita_i -= J[4 * pId + 1].dot(AA[idx1 * 2 + 1]);
			if (idx2 != -1)
			{
				ita_i -= J[4 * pId + 2].dot(AA[idx2 * 2]);
				ita_i -= J[4 * pId + 3].dot(AA[idx2 * 2 + 1]);
			}
		}

		if (d[pId] > EPSILON)
		{
			Real delta_lambda = ita_i / d[pId];
			delta_lambda *= 0.2;

			//printf("delta_lambda = %.3lf\n", delta_lambda);

			if (nbq[pId].constraint_type == constraint_collision || nbq[pId].constraint_type == constraint_boundary) //	PROJECTION!!!!
			{
				Real lambda_new = lambda[pId] + delta_lambda;
				if (lambda_new < 0) lambda_new = 0;

				Real mass_i = mass[idx1];
				if (idx2 != -1)
					mass_i += mass[idx2];

				if (lambda_new > 25 * (mass_i / 0.1)) lambda_new = 25 * (mass_i / 0.1);
				delta_lambda = lambda_new - lambda[pId];
			}

			if (nbq[pId].constraint_type == constraint_friction) //	PROJECTION!!!!
			{
				Real lambda_new = lambda[pId] + delta_lambda;
				Real mass_i = mass[idx1];
				if (idx2 != -1)
					mass_i += mass[idx2];

				//if ((lambda_new) > 15 * (mass_i)) lambda_new = 15 * (mass_i);
				//if ((lambda_new) < -15 * (mass_i)) lambda_new = -15 * (mass_i);
				delta_lambda = lambda_new - lambda[pId];
			}

			lambda[pId] += delta_lambda;

			//printf("inside iteration: %d %d %.5lf   %.5lf\n", idx1, idx2, nbq[pId].s4, delta_lambda);

			atomicAdd(&AA[idx1 * 2][0], B[4 * pId][0] * delta_lambda);
			atomicAdd(&AA[idx1 * 2][1], B[4 * pId][1] * delta_lambda);
			atomicAdd(&AA[idx1 * 2][2], B[4 * pId][2] * delta_lambda);

			atomicAdd(&AA[idx1 * 2 + 1][0], B[4 * pId + 1][0] * delta_lambda);
			atomicAdd(&AA[idx1 * 2 + 1][1], B[4 * pId + 1][1] * delta_lambda);
			atomicAdd(&AA[idx1 * 2 + 1][2], B[4 * pId + 1][2] * delta_lambda);

			if (idx2 != -1)
			{
				atomicAdd(&AA[idx2 * 2][0], B[4 * pId + 2][0] * delta_lambda);
				atomicAdd(&AA[idx2 * 2][1], B[4 * pId + 2][1] * delta_lambda);
				atomicAdd(&AA[idx2 * 2][2], B[4 * pId + 2][2] * delta_lambda);

				atomicAdd(&AA[idx2 * 2 + 1][0], B[4 * pId + 3][0] * delta_lambda);
				atomicAdd(&AA[idx2 * 2 + 1][1], B[4 * pId + 3][1] * delta_lambda);
				atomicAdd(&AA[idx2 * 2 + 1][2], B[4 * pId + 3][2] * delta_lambda);

				//AA[idx2 * 2] += B[4 * pId + 2] * delta_lambda;
				//AA[idx2 * 2 + 1] += B[4 * pId + 3] * delta_lambda;
			}
		}

	}
	

	template <typename Coord, typename Matrix> /* FOR TEST */
	__global__ void SetupContactPairs(
		DArray<NeighborConstraints> nbq,
		DArray<Coord> center_init,
		DArray<Coord> center_now,
		DArray<Matrix> rotation)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= nbq.size()) return;

		if(nbq[pId].constraint_type == constraint_distance)
		{ 
			int idx1 = nbq[pId].idx1;
			int idx2 = nbq[pId].idx2;
			Coord offset1 = nbq[pId].v1 - center_init[idx1];
			nbq[pId].pos1 = center_now[idx1] + rotation[idx1] * offset1;


			Coord offset2 = nbq[pId].v2 - center_init[idx2];
			nbq[pId].pos2 = center_now[idx2] + rotation[idx2] * offset2;
		}
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::solve_constraint()
	{
		int size_constraints = mAllConstraints.size();

		if (size_constraints == 0) return;

		for (int it = 0; it < 100; it++)
		{
			// todo : project gs
			uint pDims = cudaGridSize(size_constraints, BLOCK_SIZE);
			TakeOneJacobiIteration << <pDims, BLOCK_SIZE >> > (
				mLambda,
				mAccel,
				mD,
				mJ,
				mB,
				mEta,
				this->currentMass()->getData(),
				mAllConstraints
				);

			cuSynchronize();
		}
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::update_position_rotation(Real dt)
	{
		uint pDims = cudaGridSize(currentCenter()->getElementCount(), BLOCK_SIZE);
		
		RB_update_velocity << <pDims, BLOCK_SIZE >> > (
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			mAccel,
			dt);

		RB_update_state << <pDims, BLOCK_SIZE >> > (
			this->currentCenter()->getData(),
			this->currentRigidRotation()->getData(),
			this->currentRotation()->getData(),
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			this->currentInertia()->getData(),
			m_inertia_init,
			dt);
	}
	
	__global__ void RB_update_offset(
		DArray<NeighborConstraints> nbq,
		int offset
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= nbq.size()) return;
		if(nbq[pId].idx1 != -1)
			nbq[pId].idx1 += offset;
		if(nbq[pId].idx2 != -1)
			nbq[pId].idx2 += offset;
	}

	template <typename Coord>
	__global__ void SetupContactsWithBoundary(
		DArray<Sphere3D> sphere,
		DArray<Box3D> box,
		DArray<Tet3D> tet,
		DArray<int> count,
		DArray<NeighborConstraints> nbq,
		Coord hi,
		Coord lo,
		int start_sphere,
		int start_box,
		int start_tet,
		int start_segment)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= sphere.size() + box.size()) return;
		
		if (pId < start_box && pId >= start_sphere)//sphere
		{

		}
		else if (pId >= start_box && pId < start_tet)//box
		{
			//int idx = pId - start_box;
			int cnt = 0;
			int start_i = count[pId];
			Coord center = box[pId - start_box].center;
			Coord u = box[pId - start_box].u;
			Coord v = box[pId - start_box].v;
			Coord w = box[pId - start_box].w;
			Coord extent = box[pId - start_box].extent;
			Point3D p[8];
			p[0] = Point3D(center - u * extent[0] - v * extent[1] - w * extent[2]);
			p[1] = Point3D(center - u * extent[0] - v * extent[1] + w * extent[2]);
			p[2] = Point3D(center - u * extent[0] + v * extent[1] - w * extent[2]);
			p[3] = Point3D(center - u * extent[0] + v * extent[1] + w * extent[2]);
			p[4] = Point3D(center + u * extent[0] - v * extent[1] - w * extent[2]);
			p[5] = Point3D(center + u * extent[0] - v * extent[1] + w * extent[2]);
			p[6] = Point3D(center + u * extent[0] + v * extent[1] - w * extent[2]);
			p[7] = Point3D(center + u * extent[0] + v * extent[1] + w * extent[2]);
			bool c1, c2, c3, c4, c5, c6;
			c1 = c2 = c3 = c4 = c5 = c6 = true;
			for (int i = 0; i < 8; i++)
			{
				Coord pos = p[i].origin;
				if (pos[0] > hi[0] && c1)
				{
					c1 = true;
					nbq[cnt + start_i].idx1 = pId;
					nbq[cnt + start_i].idx2 = -1;
					nbq[cnt + start_i].normal1 = Coord(-1,0,0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].constraint_type = constraint_boundary;
					nbq[cnt + start_i].inter_distance = pos[0] - hi[0];
					cnt++;
				}
				if (pos[1] > hi[1] && c2)
				{
					c2 = true;
					nbq[cnt + start_i].idx1 = pId;
					nbq[cnt + start_i].idx2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].constraint_type = constraint_boundary;
					nbq[cnt + start_i].inter_distance = pos[1] - hi[1];
					cnt++;
				}
				if (pos[2] > hi[2] && c3)
				{
					c3 = true;
					nbq[cnt + start_i].idx1 = pId;
					nbq[cnt + start_i].idx2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].constraint_type = constraint_boundary;
					nbq[cnt + start_i].inter_distance = pos[2] - hi[2];
					cnt++;
				}
				if (pos[0] < lo[0] && c4)
				{
					c4 = true;
					nbq[cnt + start_i].idx1 = pId;
					nbq[cnt + start_i].idx2 = -1;
					nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].constraint_type = constraint_boundary;
					nbq[cnt + start_i].inter_distance = lo[0] - pos[0];
					cnt++;
				}
				if (pos[1] < lo[1] && c5)
				{
					c5 = true;
					nbq[cnt + start_i].idx1 = pId;
					nbq[cnt + start_i].idx2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].constraint_type = constraint_boundary;
					nbq[cnt + start_i].inter_distance = lo[1] - pos[1];
					cnt++;
				}
				if (pos[2] < lo[2] && c6)
				{
					c6 = true;
					nbq[cnt + start_i].idx1 = pId;
					nbq[cnt + start_i].idx2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].constraint_type = constraint_boundary;
					nbq[cnt + start_i].inter_distance = lo[2] - pos[2];
					cnt++;
				}

			}

		}
		else if (pId >= start_tet && pId < start_segment) // tets
		{
		}
		else//segments 
		{}
	}

	//template <typename Coord>
	__global__ void SetupFrictionConstraints(
		DArray<NeighborConstraints> nbq,
		int contact_size)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= contact_size) return;

		Coord3D n = nbq[pId].normal1;
		n /= n.norm();

		Coord3D n1, n2;
		if (abs(n[1]) > EPSILON || abs(n[2]) > EPSILON)
		{
			n1 = Coord3D(0, n[2], -n[1]);
			n1 /= n1.norm();
			n2 = n1.cross(n);
			n2 /= n2.norm();
		}
		else if(abs(n[0]) > EPSILON)
		{
			n1 = Coord3D(n[2], 0, -n[0]);
			n1 /= n1.norm();
			n2 = n1.cross(n);
			n2 /= n2.norm();
		}

		nbq[pId * 2 + contact_size] = nbq[pId];
		nbq[pId * 2 + contact_size].constraint_type = constraint_friction;
		nbq[pId * 2 + contact_size].normal1 = n1;
		nbq[pId * 2 + 1 + contact_size] = nbq[pId];
		nbq[pId * 2 + 1 + contact_size].constraint_type = constraint_friction;
		nbq[pId * 2 + 1 + contact_size].normal1 = n2;
	}

	template <typename Coord>
	__global__ void CountContactsWithBoundary(
		DArray<Sphere3D> sphere,
		DArray<Box3D> box,
		DArray<Tet3D> tet,
		DArray<int> count,
		Coord hi,
		Coord lo,
		int start_sphere,
		int start_box,
		int start_tet,
		int start_segment)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= sphere.size() + box.size()) return;

		if (pId < start_box && pId >= start_sphere)//sphere
		{

		}
		else if (pId >= start_box && pId < start_tet)//box
		{
			//int idx = pId - start_box;
			int cnt = 0;
//				int start_i;
			Coord center = box[pId - start_box].center;
			Coord u = box[pId - start_box].u;
			Coord v = box[pId - start_box].v;
			Coord w = box[pId - start_box].w;
			Coord extent = box[pId - start_box].extent;
			Point3D p[8];
			p[0] = Point3D(center - u * extent[0] - v * extent[1] - w * extent[2]);
			p[1] = Point3D(center - u * extent[0] - v * extent[1] + w * extent[2]);
			p[2] = Point3D(center - u * extent[0] + v * extent[1] - w * extent[2]);
			p[3] = Point3D(center - u * extent[0] + v * extent[1] + w * extent[2]);
			p[4] = Point3D(center + u * extent[0] - v * extent[1] - w * extent[2]);
			p[5] = Point3D(center + u * extent[0] - v * extent[1] + w * extent[2]);
			p[6] = Point3D(center + u * extent[0] + v * extent[1] - w * extent[2]);
			p[7] = Point3D(center + u * extent[0] + v * extent[1] + w * extent[2]);
			bool c1, c2, c3, c4, c5, c6;
			c1 = c2 = c3 = c4 = c5 = c6 = true;
			for (int i = 0; i < 8; i++)
			{
				Coord pos = p[i].origin;
				if (pos[0] > hi[0] && c1)
				{
					c1 = true;
					cnt++;
				}
				if (pos[1] > hi[1] && c2)
				{
					c2 = true;
					cnt++;
				}
				if (pos[2] > hi[2] && c3)
				{
					c3 = true;
					cnt++;
				}
				if (pos[0] < lo[0] && c4)
				{
					c4 = true;
					cnt++;
				}
				if (pos[1] < lo[1] && c5)
				{
					c5 = true;
					cnt++;
				}
				if (pos[2] < lo[2] && c6)
				{
					c6 = true;
					cnt++;
				}
			}
			count[pId] = cnt;
		}
		else if (pId >= start_tet && pId < start_segment)//tets
		{
		}
		else//segments
		{}
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::detectCollisionWithBoundary()
	{
		auto discreteSet = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());
		uint totalSize = discreteSet->totalSize();

		ElementOffset offset = discreteSet->calculateElementOffset();

		int sum = 0;

		mBoundaryContactCounter.resize(discreteSet->totalSize());
		mBoundaryContactCounter.reset();
		if (discreteSet->totalSize() > 0)
		{
			cuExecute(totalSize,
				CountContactsWithBoundary,
				discreteSet->getSpheres(),
				discreteSet->getBoxes(),
				discreteSet->getTets(),
				mBoundaryContactCounter,
				hi,
				lo,
				0,
				offset.boxOffset,
				offset.tetOffset,
				offset.segOffset);

			sum += m_reduce.accumulate(mBoundaryContactCounter.begin(), mBoundaryContactCounter.size());
			m_scan.exclusive(mBoundaryContactCounter, true);

			mBoundaryContacts.resize(sum);

			if (sum > 0) {
				cuExecute(totalSize,
					SetupContactsWithBoundary,
					discreteSet->getSpheres(),
					discreteSet->getBoxes(),
					discreteSet->getTets(),
					mBoundaryContactCounter,
					mBoundaryContacts,
					hi,
					lo,
					0,
					offset.boxOffset,
					offset.tetOffset,
					offset.segOffset);
			}
		}
		else
			mBoundaryContacts.resize(0);

		if (have_mesh_boundary)
		{

			
		}
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::init_friction()
	{




	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::init_jacobi(Real dt)
	{
		auto topo = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());

		mElementQuery->update();

		auto& contacts = mElementQuery->outContacts()->getData();

		detectCollisionWithBoundary();

		int sizeOfContacts = mBoundaryContacts.size();
		sizeOfContacts += contacts.size();

		int sizeOfConstraints = sizeOfContacts;
		if (have_friction)
		{
			sizeOfConstraints += 2 * mBoundaryContacts.size();
			if (topo->totalSize() > 0)
				sizeOfConstraints += 2 * contacts.size();
		}

		mAllConstraints.resize(sizeOfConstraints);
		
		mJ.resize(4 * sizeOfConstraints);
		mB.resize(4 * sizeOfConstraints);
		mAccel.resize(currentCenter()->getElementCount() * 2);
		mD.resize(sizeOfConstraints);
		mEta.resize(sizeOfConstraints);
		mLambda.resize(sizeOfConstraints);

		mJ.reset();
		mB.reset();
		mD.reset();
		mEta.reset();
		mAccel.reset();
		mLambda.reset();

//		mass_eq.reset();

		if (sizeOfConstraints == 0) return;

		if (topo->totalSize() > 0 && contacts.size() > 0)
			mAllConstraints.assign(contacts, contacts.size());
		
		if (mBoundaryContacts.size() > 0)
		{
			if (topo->totalSize() > 0)
			{ 
				if (!have_mesh)
					mAllConstraints.assign(mBoundaryContacts, mBoundaryContacts.size(), contacts.size(), 0);
			}
			else
			{
				mAllConstraints.assign(mBoundaryContacts, mBoundaryContacts.size());
			}
		}

		if (have_friction)
		{
			cuExecute(sizeOfContacts, 
				SetupFrictionConstraints,
				mAllConstraints,
				sizeOfContacts
				);
		}
// 		uint pDims = cudaGridSize(size_constraints, BLOCK_SIZE);
// 		uint pDimsR = cudaGridSize(currentMass()->getElementCount(), BLOCK_SIZE);

		//TODO: ???
// 		if(use_new_mass)
// 		{
// 			printf("?????? USE NEW\n");
// 			mass_eq.reset();
// 			mass_buffer.reset();
// 
// 			for (int it = 0; it < 15; it++)
// 			{
// 				mass_eq.reset();
// 				RB_constrct_mass_eq << <pDims, BLOCK_SIZE >> > (
// 					currentCenter()->getData(),
// 					m_inertia.getData(),
// 					currentMass()->getData(),
// 					mJ,
// 					mB,
// 					mass_eq,
// 					mass_buffer,
// 					m_inertia.getData(),
// 					constraints_all
// 					);
// 				cuSynchronize();
// 
// 				RB_constrct_mass_eq<< <pDimsR, BLOCK_SIZE >> > (
// 					m_inertia.getData(),
// 					currentMass()->getData(),
// 					mass_eq
// 					);
// 				cuSynchronize();
// 				//Function1Pt::copy(mass_buffer, mass_eq);
// 				mass_buffer.assign(mass_eq);
// 			}
// 		}
		
		cuExecute(sizeOfConstraints,
			SetupContactPairs,
			mAllConstraints,
			center_init,
			this->currentCenter()->getData(),
			this->currentRigidRotation()->getData());

// 		cuSynchronize();
// 		if (use_new_mass)
// 		{
// 			cuExecute(size_constraints,
// 				CalculateJacobians,
// 				mJ,
// 				mB,
// 				currentCenter()->getData(),
// 				m_inertia.getData(),
// 				m_inertia.getData(),
// 				currentMass()->getData(),
// 				mass_eq,
// 				constraints_all);
// 		}
// 		else
		{ 
			cuExecute(sizeOfConstraints,
				CalculateJacobians,
				mJ,
				mB,
				this->currentCenter()->getData(),
				this->currentInertia()->getData(),
				this->currentMass()->getData(),
				mAllConstraints);
		}

		cuExecute(sizeOfConstraints,
			CalculateDiagonals,
			mD,
			mJ,
			mB);

		cuExecute(sizeOfConstraints, 
			CalculateEta,
			mEta,
			this->currentVelocity()->getData(),
			this->currentAngularVelocity()->getData(),
			mJ,
			this->currentMass()->getData(),
			mAllConstraints,
			dt);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::updateStates()
	{
		Real dt = this->varTimeStep()->getData();
		//construct j
		init_jacobi(dt);
		for (int i = 0; i < 15; i++)
		{
			int size_constraints = mAllConstraints.size();
			if (size_constraints == 0) return;
			uint pDims = cudaGridSize(size_constraints, BLOCK_SIZE);

			TakeOneJacobiIteration << <pDims, BLOCK_SIZE >> > (
				mLambda,
				mAccel,
				mD,
				mJ,
				mB,
				mEta,
				currentMass()->getData(),
				mAllConstraints);
		}

		update_position_rotation(dt);
	}

	template<typename TDataType>
	void RigidBodySystem<TDataType>::updateTopology()
	{
		auto discreteSet = TypeInfo::cast<DiscreteElements<DataType3f>>(this->currentTopology()->getDataPtr());

		ElementOffset offset = discreteSet->calculateElementOffset();

		cuExecute(mDeviceBoxes.size(),
			UpdateBoxes,
			discreteSet->getBoxes(),
			mDeviceBoxes,
			this->currentCenter()->getData(),
			this->currentRigidRotation()->getData(),
			offset.boxOffset);

		cuExecute(mDeviceBoxes.size(),
			UpdateSpheres,
			discreteSet->getSpheres(),
			this->currentCenter()->getData(),
			0);

		cuExecute(mDeviceTets.size(),
			UpdateTets,
			discreteSet->getTets(),
			mDeviceTets,
			this->currentCenter()->getData(),
			this->currentRigidRotation()->getData(),
			offset.tetOffset);
	}

	DEFINE_CLASS(RigidBodySystem);
}